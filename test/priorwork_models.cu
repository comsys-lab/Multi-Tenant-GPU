#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <unistd.h>
#include <string.h>
#include <math.h>
#include <time.h>

#include "Kernel.cu"

#define INPUT_SIZE 224*224*3


/* Function to Read Alexnet Input Parameters */
extern "C"{
void read_parameter(const char *pFileName,float *layer_parameters)
{
	FILE *fp = fopen(pFileName, "rb");
	int count = 0;
	double temp_num;
	//printf(" File FOUND : %s\n",pFileName);
	while(fscanf(fp, "%lf", &temp_num) == 1){
		layer_parameters[count] = temp_num;
		count++;
	}
	//printf("Final Count : %d\n", count);
	fclose(fp);
}

void host2gpu_alexnet(float **Alex_Layer1_Neurons,float **Alex_Layer2_Neurons,float **Alex_Layer3_Neurons,float **Alex_Layer4_Neurons,
					float **Alex_Layer5_Neurons,float **Alex_Layer6_Neurons,float **Alex_Layer7_Neurons,float **Alex_Layer8_Neurons,
                    float **Alex_Layer1_bias,float **Alex_Layer2_bias,float **Alex_Layer3_bias,float **Alex_Layer4_bias,
                    float **Alex_Layer5_bias,float **Alex_Layer6_bias,float **Alex_Layer7_bias,float **Alex_Layer8_bias,
                    float **Alex_Layer1_Weights,float **Alex_Layer2_Weights,float **Alex_Layer3_Weights,float **Alex_Layer4_Weights,
                    float **Alex_Layer5_Weights,float **Alex_Layer6_Weights,float **Alex_Layer7_Weights,float **Alex_Layer8_Weights,
                    float **Alex_Layer1_pool,float **Alex_Layer2_pool,float **Alex_Layer5_pool,
					float **Alex_Layer1_norm,float **Alex_Layer2_norm,float **Alex_Result_Neurons)
{

	float *Alex_Layer1_Neurons_CPU = (float*) malloc (INPUT_SIZE * sizeof(float));
	read_parameter("data_alexnet/input_cat1.txt", Alex_Layer1_Neurons_CPU);

	float *Alex_Layer1_bias_CPU = (float*) malloc (64 * sizeof(float));
	float *Alex_Layer2_bias_CPU = (float*) malloc (192 * sizeof(float));
	float *Alex_Layer3_bias_CPU = (float*) malloc (384 * sizeof(float));
	float *Alex_Layer4_bias_CPU = (float*) malloc (256 * sizeof(float));
	float *Alex_Layer5_bias_CPU = (float*) malloc (256 * sizeof(float));
	float *Alex_Layer6_bias_CPU = (float*) malloc (4096 * sizeof(float));
	float *Alex_Layer7_bias_CPU = (float*) malloc (4096 * sizeof(float));
	float *Alex_Layer8_bias_CPU = (float*) malloc (1000 * sizeof(float));

	float *Alex_Layer1_Weights_CPU = (float*) malloc (64*11*11*3 * sizeof(float));
	float *Alex_Layer2_Weights_CPU = (float*) malloc (192*5*5*64 * sizeof(float));
	float *Alex_Layer3_Weights_CPU = (float*) malloc (384*3*3*192 * sizeof(float));
	float *Alex_Layer4_Weights_CPU = (float*) malloc (256*3*3*384 * sizeof(float));
	float *Alex_Layer5_Weights_CPU = (float*) malloc (256*3*3*256 * sizeof(float));
	float *Alex_Layer6_Weights_CPU = (float*) malloc (4096*256*6*6 * sizeof(float));
	float *Alex_Layer7_Weights_CPU = (float*) malloc (4096*4096 * sizeof(float));
	float *Alex_Layer8_Weights_CPU = (float*) malloc (1000*4096 * sizeof(float));

	read_parameter("data_alexnet/bias1.txt", Alex_Layer1_bias_CPU);
	read_parameter("data_alexnet/bias2.txt", Alex_Layer2_bias_CPU);
	read_parameter("data_alexnet/bias3.txt", Alex_Layer3_bias_CPU);
	read_parameter("data_alexnet/bias4.txt", Alex_Layer4_bias_CPU);
	read_parameter("data_alexnet/bias5.txt", Alex_Layer5_bias_CPU);
	read_parameter("data_alexnet/bias6.txt", Alex_Layer6_bias_CPU);
	read_parameter("data_alexnet/bias7.txt", Alex_Layer7_bias_CPU);
	read_parameter("data_alexnet/bias8.txt", Alex_Layer8_bias_CPU);

	read_parameter("data_alexnet/conv1.txt", Alex_Layer1_Weights_CPU);
	read_parameter("data_alexnet/conv2.txt", Alex_Layer2_Weights_CPU);
	read_parameter("data_alexnet/conv3.txt", Alex_Layer3_Weights_CPU);
	read_parameter("data_alexnet/conv4.txt", Alex_Layer4_Weights_CPU);
	read_parameter("data_alexnet/conv5.txt", Alex_Layer5_Weights_CPU);
	read_parameter("data_alexnet/fc6.txt", Alex_Layer6_Weights_CPU);
	read_parameter("data_alexnet/fc7.txt", Alex_Layer7_Weights_CPU);
	read_parameter("data_alexnet/fc8.txt", Alex_Layer8_Weights_CPU);

    float *Alex_Layer1_Neurons_data;
	float *Alex_Layer1_bias_data, *Alex_Layer2_bias_data, *Alex_Layer3_bias_data, *Alex_Layer4_bias_data, 
			*Alex_Layer5_bias_data, *Alex_Layer6_bias_data, *Alex_Layer7_bias_data, *Alex_Layer8_bias_data;
	float *Alex_Layer1_Weights_data, *Alex_Layer2_Weights_data, *Alex_Layer3_Weights_data, *Alex_Layer4_Weights_data,
			*Alex_Layer5_Weights_data, *Alex_Layer6_Weights_data, *Alex_Layer7_Weights_data, *Alex_Layer8_Weights_data;

	hipMalloc((void**) &Alex_Layer1_Neurons_data, INPUT_SIZE * sizeof(float)); //224*224*3
	hipMalloc((void**) &Alex_Layer1_bias_data, 64 * sizeof(float)); //64
	hipMalloc((void**) &Alex_Layer1_Weights_data, (64*11*11*3) * sizeof(float)); //64*11*11*3 = 23232
	hipMalloc((void**) &Alex_Layer2_bias_data, 192 * sizeof(float)); //192
	hipMalloc((void**) &Alex_Layer2_Weights_data, (192*5*5*64) * sizeof(float)); //192*5*5*64 = 307200
	hipMalloc((void**) &Alex_Layer3_bias_data, 384 * sizeof(float)); //384
	hipMalloc((void**) &Alex_Layer3_Weights_data, (384*3*3*192) * sizeof(float)); //384*3*3*192 = 663552
	hipMalloc((void**) &Alex_Layer4_bias_data, 256 * sizeof(float)); //256
	hipMalloc((void**) &Alex_Layer4_Weights_data, (256*3*3*384) * sizeof(float)); //256*3*3*384 = 884736
	hipMalloc((void**) &Alex_Layer5_bias_data, 256 * sizeof(float)); //256
	hipMalloc((void**) &Alex_Layer5_Weights_data, (256*3*3*256) * sizeof(float)); //256*3*3*256 = 442368
	hipMalloc((void**) &Alex_Layer6_bias_data, 4096 * sizeof(float)); //4096
	hipMalloc((void**) &Alex_Layer6_Weights_data, (4096*256*6*6) * sizeof(float)); //4096*256*6*6 = 37748736
	hipMalloc((void**) &Alex_Layer7_bias_data, 4096 * sizeof(float)); //4096
	hipMalloc((void**) &Alex_Layer7_Weights_data, (4096*4096) * sizeof(float)); //4096*4096 = 16777216
	hipMalloc((void**) &Alex_Layer8_bias_data, 1000 * sizeof(float)); //1000
	hipMalloc((void**) &Alex_Layer8_Weights_data, (1000*4096) * sizeof(float)); //1000*4096 = 4096000
	
	hipMemcpy(Alex_Layer1_Neurons_data, Alex_Layer1_Neurons_CPU, INPUT_SIZE * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(Alex_Layer1_bias_data, Alex_Layer1_bias_CPU, 64 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(Alex_Layer1_Weights_data, Alex_Layer1_Weights_CPU, (64*11*11*3) * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(Alex_Layer2_bias_data, Alex_Layer2_bias_CPU, 192 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(Alex_Layer2_Weights_data, Alex_Layer2_Weights_CPU, (192*5*5*64) * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(Alex_Layer3_bias_data, Alex_Layer3_bias_CPU, 384 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(Alex_Layer3_Weights_data, Alex_Layer3_Weights_CPU, (384*3*3*192) * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(Alex_Layer4_bias_data, Alex_Layer4_bias_CPU, 256 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(Alex_Layer4_Weights_data, Alex_Layer4_Weights_CPU, (256*3*3*384) * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(Alex_Layer5_bias_data, Alex_Layer5_bias_CPU, 256 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(Alex_Layer5_Weights_data, Alex_Layer5_Weights_CPU, (256*3*3*256) * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(Alex_Layer6_bias_data, Alex_Layer6_bias_CPU, 4096 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(Alex_Layer6_Weights_data, Alex_Layer6_Weights_CPU, (4096*256*6*6) * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(Alex_Layer7_bias_data, Alex_Layer7_bias_CPU, 4096 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(Alex_Layer7_Weights_data, Alex_Layer7_Weights_CPU, (4096*4096) * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(Alex_Layer8_bias_data, Alex_Layer8_bias_CPU, 1000 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(Alex_Layer8_Weights_data, Alex_Layer8_Weights_CPU, (1000*4096) * sizeof(float), hipMemcpyHostToDevice);

	*Alex_Layer1_Neurons = Alex_Layer1_Neurons_data;

	*Alex_Layer1_bias = Alex_Layer1_bias_data;
	*Alex_Layer2_bias = Alex_Layer2_bias_data;
	*Alex_Layer3_bias = Alex_Layer3_bias_data;
	*Alex_Layer4_bias = Alex_Layer4_bias_data;
	*Alex_Layer5_bias = Alex_Layer5_bias_data;
	*Alex_Layer6_bias = Alex_Layer6_bias_data;
	*Alex_Layer7_bias = Alex_Layer7_bias_data;
	*Alex_Layer8_bias = Alex_Layer8_bias_data;

	*Alex_Layer1_Weights = Alex_Layer1_Weights_data;
	*Alex_Layer2_Weights = Alex_Layer2_Weights_data;
	*Alex_Layer3_Weights = Alex_Layer3_Weights_data;
	*Alex_Layer4_Weights = Alex_Layer4_Weights_data;
	*Alex_Layer5_Weights = Alex_Layer5_Weights_data;
	*Alex_Layer6_Weights = Alex_Layer6_Weights_data;
	*Alex_Layer7_Weights = Alex_Layer7_Weights_data;
	*Alex_Layer8_Weights = Alex_Layer8_Weights_data;

	free(Alex_Layer1_Neurons_CPU);

	free(Alex_Layer1_bias_CPU);
	free(Alex_Layer2_bias_CPU);
	free(Alex_Layer3_bias_CPU);
	free(Alex_Layer4_bias_CPU);
	free(Alex_Layer5_bias_CPU);
	free(Alex_Layer6_bias_CPU);
	free(Alex_Layer7_bias_CPU);
	free(Alex_Layer8_bias_CPU);

    free(Alex_Layer1_Weights_CPU);
    free(Alex_Layer2_Weights_CPU);
    free(Alex_Layer3_Weights_CPU);
    free(Alex_Layer4_Weights_CPU);
    free(Alex_Layer5_Weights_CPU);
    free(Alex_Layer6_Weights_CPU);
    free(Alex_Layer7_Weights_CPU);
    free(Alex_Layer8_Weights_CPU);

	float *Alex_Layer1_norm_data; 
	hipMalloc((void**) &Alex_Layer1_norm_data, (64*55*55) * sizeof(float)); //64*55*55 
	*Alex_Layer1_norm = Alex_Layer1_norm_data;

	float *Alex_Layer1_pool_data;
    hipMalloc((void**) &Alex_Layer1_pool_data, (64*55*55) * sizeof(float)); //64*55*55
	*Alex_Layer1_pool = Alex_Layer1_pool_data;

	float *Alex_Layer2_Neurons_data;
	hipMalloc((void**) &Alex_Layer2_Neurons_data, (64*27*27) * sizeof(float)); //64*27*27
	*Alex_Layer2_Neurons = Alex_Layer2_Neurons_data;

	float *Alex_Layer2_norm_data;
	hipMalloc((void**) &Alex_Layer2_norm_data, (192*27*27) * sizeof(float)); //192*27*27
	*Alex_Layer2_norm = Alex_Layer2_norm_data;

	float *Alex_Layer2_pool_data;
    hipMalloc((void**) &Alex_Layer2_pool_data, (192*27*27) * sizeof(float)); //192*27*27
	*Alex_Layer2_pool = Alex_Layer2_pool_data;

	float *Alex_Layer3_Neurons_data;
    hipMalloc((void**) &Alex_Layer3_Neurons_data, (192*13*13) * sizeof(float)); //192*13*13
	*Alex_Layer3_Neurons = Alex_Layer3_Neurons_data;

	float *Alex_Layer4_Neurons_data;
    hipMalloc((void**) &Alex_Layer4_Neurons_data, (384*13*13) * sizeof(float)); //384*13*13
	*Alex_Layer4_Neurons = Alex_Layer4_Neurons_data;

	float *Alex_Layer5_Neurons_data;
	hipMalloc((void**) &Alex_Layer5_Neurons_data, (256*13*13) * sizeof(float)); //256*13*13
	*Alex_Layer5_Neurons = Alex_Layer5_Neurons_data;

	float *Alex_Layer5_pool_data;
	hipMalloc((void**) &Alex_Layer5_pool_data, (256*13*13) * sizeof(float)); //256*13*13
	*Alex_Layer5_pool = Alex_Layer5_pool_data;

	float *Alex_Layer6_Neurons_data;
	hipMalloc((void**) &Alex_Layer6_Neurons_data, (256*6*6) * sizeof(float)); //256*6*6
	*Alex_Layer6_Neurons = Alex_Layer6_Neurons_data;

	float *Alex_Layer7_Neurons_data;
	hipMalloc((void**) &Alex_Layer7_Neurons_data, 4096 * sizeof(float)); //4096
	*Alex_Layer7_Neurons = Alex_Layer7_Neurons_data;

	float *Alex_Layer8_Neurons_data;
	hipMalloc((void**) &Alex_Layer8_Neurons_data, 4096 * sizeof(float)); //4096
	*Alex_Layer8_Neurons = Alex_Layer8_Neurons_data;

	float *Alex_Result_Neurons_data;
	hipMalloc((void**) &Alex_Result_Neurons_data, 1000 * sizeof(float)); //1000
	*Alex_Result_Neurons = Alex_Result_Neurons_data;
}

// float start_time()
// {
// 	hipEvent_t start;
// 	hipEventCreate(&start);
// 	hipEventRecord(start,0);

// 	float host_start;
//     hipMemcpy(&host_start, &start, sizeof(float), hipMemcpyDeviceToHost);
//	return host_start;
// 	hipEventDestroy(start);
// }

// float end_time()
// {
// 	hipEvent_t end;
// 	hipEventCreate(&end);
// 	hipEventRecord(end,0);

// 	float host_end;
//    hipMemcpy(&host_end, &end, sizeof(float), hipMemcpyDeviceToHost);
// 	return host_end;
// 	hipEventDestroy(end);
// }
void loop_func()
{
	while(1)
		continue;	
}

void alex_first_conv(float *Alex_Layer1_bias,float *Alex_Layer1_Neurons,float *Alex_Layer1_Weights,float *Alex_Layer1_norm)
{
    dim3 Layer1_Block(64,5,5);
	dim3 Layer1_Thread(11,11);
	first<<<Layer1_Block,Layer1_Thread>>>(Alex_Layer1_bias,Alex_Layer1_Neurons,Alex_Layer1_Weights,Alex_Layer1_norm,224,55,4,2,11,3,true,true);
}

void alex_fisrt_norm(float *Alex_Layer1_norm,float *Alex_Layer1_pool)
{
   	dim3 Norm11_Block(64,5,5);
	dim3 Norm11_Thread(11,11);
	norm<<<Norm11_Block,Norm11_Thread>>>(Alex_Layer1_norm,Alex_Layer1_pool,0.0001,0.75,5,55); 
}

void alex_first_pool(float *Alex_Layer1_pool,float *Alex_Layer2_Neurons)
{
    dim3 Pool1_Block(64,1,1);
	dim3 Pool1_Thread(27,27);
	max<<<Pool1_Block,Pool1_Thread>>>(Alex_Layer1_pool,Alex_Layer2_Neurons,55,27,2,0,3);
}

void alex_second_conv(float *Alex_Layer2_bias,float *Alex_Layer2_Neurons,float *Alex_Layer2_Weights,float *Alex_Layer2_norm)
{
    dim3 Layer2_Block(192,1,1);
	dim3 Layer2_Thread(27,27); 
	conv<<<Layer2_Block,Layer2_Thread>>>(Alex_Layer2_bias,Alex_Layer2_Neurons,Alex_Layer2_Weights,Alex_Layer2_norm,27,27,1,2,5,64,true,true);
}

void alex_second_norm(float *Alex_Layer2_norm,float *Alex_Layer2_pool)
{
    dim3 Norm2_Block(192,1,1);
	dim3 Norm2_Thread(27,27);
	norm<<<Norm2_Block,Norm2_Thread>>>(Alex_Layer2_norm,Alex_Layer2_pool,0.0001,0.75,5,27);
}

void alex_second_pool(float *Alex_Layer2_pool,float *Alex_Layer3_Neurons)
{
    dim3 Pool2_Block(192,1,1);
	dim3 Pool2_Thread(13,13);
	max<<<Pool2_Block,Pool2_Thread>>>(Alex_Layer2_pool,Alex_Layer3_Neurons,27,13,2,0,3);
}

void alex_third_conv(float *Alex_Layer3_bias,float *Alex_Layer3_Neurons,float *Alex_Layer3_Weights,float *Alex_Layer4_Neurons)
{
	dim3 Layer3_Block(384,1,1);
	dim3 Layer3_Thread(13,13); 
	conv<<<Layer3_Block,Layer3_Thread>>>(Alex_Layer3_bias,Alex_Layer3_Neurons,Alex_Layer3_Weights,Alex_Layer4_Neurons,13,13,1,1,3,192,true,true);
}

void alex_fourth_conv(float *Alex_Layer4_bias,float *Alex_Layer4_Neurons,float *Alex_Layer4_Weights,float *Alex_Layer5_Neurons)
{
    dim3 Layer4_Block(256,1,1);
	dim3 Layer4_Thread(13,13); 
	conv<<<Layer4_Block,Layer4_Thread>>>(Alex_Layer4_bias,Alex_Layer4_Neurons,Alex_Layer4_Weights,Alex_Layer5_Neurons,13,13,1,1,3,384,true,true);
}

void alex_fifth_conv(float *Alex_Layer5_bias,float *Alex_Layer5_Neurons,float *Alex_Layer5_Weights,float *Alex_Layer5_pool)
{
    dim3 Layer5_Block(256,1,1);
	dim3 Layer5_Thread(13,13); 
	conv<<<Layer5_Block,Layer5_Thread>>>(Alex_Layer5_bias,Alex_Layer5_Neurons,Alex_Layer5_Weights,Alex_Layer5_pool,13,13,1,1,3,256,true,true);
}

void alex_fifth_pool(float *Alex_Layer5_pool,float *Alex_Layer6_Neurons)
{
	dim3 Pool3_Block(256,1,1);
	dim3 Pool3_Thread(6,6);
	max<<<Pool3_Block,Pool3_Thread>>>(Alex_Layer5_pool,Alex_Layer6_Neurons,13,6,2,0,3);
}

void alex_first_fc(float *Alex_Layer6_bias,float *Alex_Layer6_Neurons,float *Alex_Layer6_Weights,float *Alex_Layer7_Neurons)
{	
	dim3 Layer6_Block(4096,1,1);
	dim3 Layer6_Thread(1,1);
	fc<<<Layer6_Block,Layer6_Thread>>>(Alex_Layer6_bias,Alex_Layer6_Neurons,Alex_Layer6_Weights,Alex_Layer7_Neurons,(6*6*256),true);

}

void alex_second_fc(float *Alex_Layer7_bias,float *Alex_Layer7_Neurons,float *Alex_Layer7_Weights,float *Alex_Layer8_Neurons)
{
	dim3 Layer7_Block(4096,1,1);
	dim3 Layer7_Thread(1,1);
	fc<<<Layer7_Block,Layer7_Thread>>>(Alex_Layer7_bias,Alex_Layer7_Neurons,Alex_Layer7_Weights,Alex_Layer8_Neurons,4096,true);
}

void alex_third_fc(float *Alex_Layer8_bias,float *Alex_Layer8_Neurons,float *Alex_Layer8_Weights,float *Alex_Result_Neurons)
{


    dim3 Layer8_Block(1000,1,1);
	dim3 Layer8_Thread(1,1);
	fc<<<Layer8_Block,Layer8_Thread>>>(Alex_Layer8_bias,Alex_Layer8_Neurons,Alex_Layer8_Weights,Alex_Result_Neurons,4096,false);

    float *Alex_Result_Neurons_CPU = (float *) malloc ((1000) * sizeof(float));
	hipMemcpy(Alex_Result_Neurons_CPU, Alex_Result_Neurons, (1000) * sizeof(float), hipMemcpyDeviceToHost);

	float max1 = 0.0;
	int index1 = 0; 
	for(int i = 0; i < 1000; i++){
		if(max1 < Alex_Result_Neurons_CPU[i]){
			max1 = Alex_Result_Neurons_CPU[i];	
			index1 = i;
		}
	}
	
	int line_count1 = 0;
	char buffer[1000];
	FILE *list1 = fopen("imagenet1000_clsidx_to_labels.txt","rt");
	while(fgets(buffer, 1000, list1) != NULL){
		line_count1++;
		if(line_count1 == (index1+1)){
			// printf("\n---Alexnet Result---");
			// printf("\nClass ID: %d\nClass Name: %sProbability: %f\n", index1, buffer, max1);
			printf("\nAlexnet: %d, %s", index1, buffer);
			break;
		}
	}
	fclose(list1);
	
	free(Alex_Result_Neurons_CPU);
}

void free_alexnet(float *Alex_Layer1_Neurons,float *Alex_Layer2_Neurons,float *Alex_Layer3_Neurons,float *Alex_Layer4_Neurons,
					float *Alex_Layer5_Neurons,float *Alex_Layer6_Neurons,float *Alex_Layer7_Neurons,float *Alex_Layer8_Neurons,
                    float *Alex_Layer1_bias,float *Alex_Layer2_bias,float *Alex_Layer3_bias,float *Alex_Layer4_bias,
                    float *Alex_Layer5_bias,float *Alex_Layer6_bias,float *Alex_Layer7_bias,float *Alex_Layer8_bias,
                    float *Alex_Layer1_Weights,float *Alex_Layer2_Weights,float *Alex_Layer3_Weights,float *Alex_Layer4_Weights,
                    float *Alex_Layer5_Weights,float * Alex_Layer6_Weights,float *Alex_Layer7_Weights,float *Alex_Layer8_Weights,
                    float *Alex_Layer1_pool,float *Alex_Layer2_pool,float *Alex_Layer5_pool,
					float *Alex_Layer1_norm,float *Alex_Layer2_norm,float *Alex_Result_Neurons)
{
	hipFree(Alex_Layer1_Neurons);
	hipFree(Alex_Layer2_Neurons);
	hipFree(Alex_Layer3_Neurons);
	hipFree(Alex_Layer4_Neurons);
	hipFree(Alex_Layer5_Neurons);
	hipFree(Alex_Layer6_Neurons);
	hipFree(Alex_Layer7_Neurons);
	hipFree(Alex_Layer8_Neurons);

	hipFree(Alex_Layer1_bias);
	hipFree(Alex_Layer2_bias);
	hipFree(Alex_Layer3_bias);
	hipFree(Alex_Layer4_bias);
	hipFree(Alex_Layer5_bias);
	hipFree(Alex_Layer6_bias);
	hipFree(Alex_Layer7_bias);
	hipFree(Alex_Layer8_bias);

	hipFree(Alex_Layer1_Weights);
	hipFree(Alex_Layer2_Weights);
	hipFree(Alex_Layer3_Weights);
	hipFree(Alex_Layer4_Weights);
	hipFree(Alex_Layer5_Weights);
	hipFree(Alex_Layer6_Weights);
	hipFree(Alex_Layer7_Weights);
	hipFree(Alex_Layer8_Weights);

	hipFree(Alex_Layer1_pool);
	hipFree(Alex_Layer2_pool);
	hipFree(Alex_Layer5_pool);
	hipFree(Alex_Layer1_norm);
	hipFree(Alex_Layer2_norm);
	hipFree(Alex_Result_Neurons);
}

void host2gpu_resnet18(float **Res_Layer1_Neurons,float **Res_Layer2_Neurons,float **Res_Layer3_Neurons,float **Res_Layer4_Neurons,
					float **Res_Layer5_Neurons,float **Res_Layer6_Neurons,float **Res_Layer7_Neurons,float **Res_Layer8_Neurons,
					float **Res_Layer9_Neurons,float **Res_Layer10_Neurons,float **Res_Layer11_Neurons,float **Res_Layer12_Neurons,
					float **Res_Layer13_Neurons,float **Res_Layer14_Neurons,float **Res_Layer15_Neurons,float **Res_Layer16_Neurons,
					float **Res_Layer17_Neurons,float **Res_Layer18_Neurons,
                    float **Res_Layer1_Weights,float **Res_Layer2_Weights,float **Res_Layer3_Weights,float **Res_Layer4_Weights,
                    float **Res_Layer5_Weights,float **Res_Layer6_Weights,float **Res_Layer7_Weights,float **Res_Layer8_Weights,
                    float **Res_Layer9_Weights,float **Res_Layer10_Weights,float **Res_Layer11_Weights,float **Res_Layer12_Weights,
                    float **Res_Layer13_Weights,float **Res_Layer14_Weights,float **Res_Layer15_Weights,float **Res_Layer16_Weights,
                    float **Res_Layer17_Weights,float **Res_Block3_Weights,float **Res_Block4_Weights,float **Res_Block5_Weights,
                    float **Res_Layer1_Gamma,float **Res_Layer2_Gamma,float **Res_Layer3_Gamma,float **Res_Layer4_Gamma,
                    float **Res_Layer5_Gamma,float **Res_Layer6_Gamma,float **Res_Layer7_Gamma,float **Res_Layer8_Gamma,
                    float **Res_Layer9_Gamma,float **Res_Layer10_Gamma,float **Res_Layer11_Gamma,float **Res_Layer12_Gamma,
                    float **Res_Layer13_Gamma,float **Res_Layer14_Gamma,float **Res_Layer15_Gamma,float **Res_Layer16_Gamma,
                    float **Res_Layer17_Gamma,float **Res_Block3_Gamma,float **Res_Block4_Gamma,float **Res_Block5_Gamma,
                    float **Res_Layer1_Beta,float **Res_Layer2_Beta,float**Res_Layer3_Beta,float **Res_Layer4_Beta,
                    float **Res_Layer5_Beta,float **Res_Layer6_Beta,float **Res_Layer7_Beta,float **Res_Layer8_Beta,
                    float **Res_Layer9_Beta,float **Res_Layer10_Beta,float **Res_Layer11_Beta,float **Res_Layer12_Beta,
                    float **Res_Layer13_Beta,float **Res_Layer14_Beta,float **Res_Layer15_Beta,float **Res_Layer16_Beta,
                    float **Res_Layer17_Beta,float **Res_Block3_Beta,float **Res_Block4_Beta,float **Res_Block5_Beta,
                    float **Res_mean1,float **Res_mean2,float **Res_mean3,float **Res_mean4,float **Res_mean5,
                    float **Res_mean6,float **Res_mean7,float **Res_mean8,float **Res_mean9,float **Res_mean10,
                    float **Res_mean11,float **Res_mean12,float **Res_mean13,float **Res_mean14,float **Res_mean15,
                    float **Res_mean16,float **Res_mean17,float **Res_Block3_mean,float **Res_Block4_mean,float **Res_Block5_mean,
                    float **Res_var1,float **Res_var2,float **Res_var3,float **Res_var4,float **Res_var5,
                    float **Res_var6,float **Res_var7,float **Res_var8,float **Res_var9,float **Res_var10,
                    float **Res_var11,float **Res_var12,float **Res_var13,float **Res_var14,float **Res_var15,
                    float **Res_var16,float **Res_var17,float **Res_Block3_var,float **Res_Block4_var,float **Res_Block5_var,
                    float **Res_FC_bias,float **Res_FC_Weights,
					float **Res_Layer3_basic,float **Res_Layer5_basic,float **Res_Layer7_basic,float **Res_Layer9_basic,
					float **Res_Layer11_basic,float **Res_Layer13_basic,float **Res_Layer15_basic,float **Res_Layer17_basic,
					float **Res_Block3_basic,float **Res_Block4_basic,float **Res_Block5_basic,
					float **Res_Layer1_bn,float **Res_Layer2_bn,float **Res_Layer3_bn,float **Res_Layer4_bn,
					float **Res_Layer5_bn,float **Res_Layer6_bn,float **Res_Layer7_bn,float **Res_Layer8_bn,
					float **Res_Layer9_bn,float **Res_Layer10_bn,float **Res_Layer11_bn,float **Res_Layer12_bn,
					float **Res_Layer13_bn,float **Res_Layer14_bn,float **Res_Layer15_bn,float **Res_Layer16_bn,
					float **Res_Layer17_bn,float **Res_Block3_bn,float **Res_Block4_bn,float **Res_Block5_bn,
					float **Res_Layer1_pool,float **Res_FC_Neurons,float **Res_Result_Neurons)
{
	float *Res_Layer1_Neurons_CPU = (float*) malloc (INPUT_SIZE * sizeof(float));
	read_parameter("data_resnet18/input_cat.txt", Res_Layer1_Neurons_CPU);

	float *Res_Layer1_Weights_CPU = (float*) malloc ((7*7*3*64) * sizeof(float)); // = 9,408
	float *Res_Layer2_Weights_CPU = (float*) malloc ((3*3*64*64) * sizeof(float)); // = 36,864
	float *Res_Layer3_Weights_CPU = (float*) malloc ((3*3*64*64) * sizeof(float)); // = 36,864
	float *Res_Layer4_Weights_CPU = (float*) malloc ((3*3*64*64) * sizeof(float)); // = 36,864
	float *Res_Layer5_Weights_CPU = (float*) malloc ((3*3*64*64) * sizeof(float)); // = 36,864
	float *Res_Layer6_Weights_CPU = (float*) malloc ((3*3*64*128) * sizeof(float)); // = 73,728
	float *Res_Layer7_Weights_CPU = (float*) malloc ((3*3*128*128) * sizeof(float)); // = 147,456
	float *Res_Layer8_Weights_CPU = (float*) malloc ((3*3*128*128) * sizeof(float)); // = 147,456
    float *Res_Layer9_Weights_CPU = (float*) malloc ((3*3*128*128) * sizeof(float)); // = 147,456
	float *Res_Layer10_Weights_CPU = (float*) malloc ((3*3*128*256) * sizeof(float)); // = 294,912
	float *Res_Layer11_Weights_CPU = (float*) malloc ((3*3*256*256) * sizeof(float)); // = 589,824
	float *Res_Layer12_Weights_CPU = (float*) malloc ((3*3*256*256) * sizeof(float)); // = 589,824
	float *Res_Layer13_Weights_CPU = (float*) malloc ((3*3*256*256) * sizeof(float)); // = 589,824
	float *Res_Layer14_Weights_CPU = (float*) malloc ((3*3*256*512) * sizeof(float)); // = 1,179,648
	float *Res_Layer15_Weights_CPU = (float*) malloc ((3*3*512*512) * sizeof(float)); // = 2,359,296
	float *Res_Layer16_Weights_CPU = (float*) malloc ((3*3*512*512) * sizeof(float)); // = 2,359,296
	float *Res_Layer17_Weights_CPU = (float*) malloc ((3*3*512*512) * sizeof(float)); // = 2,359,296
	float *Res_Block3_Weights_CPU = (float*) malloc ((1*1*64*128) * sizeof(float)); // = 8,192
	float *Res_Block4_Weights_CPU = (float*) malloc ((1*1*128*256) * sizeof(float)); // = 32,768
	float *Res_Block5_Weights_CPU = (float*) malloc ((1*1*256*512) * sizeof(float)); // = 131,072
   
    float *Res_Layer1_Gamma_CPU = (float*) malloc (64 * sizeof(float));
	float *Res_Layer2_Gamma_CPU = (float*) malloc (64 * sizeof(float)); 
	float *Res_Layer3_Gamma_CPU = (float*) malloc (64 * sizeof(float)); 
	float *Res_Layer4_Gamma_CPU = (float*) malloc (64 * sizeof(float)); 
	float *Res_Layer5_Gamma_CPU = (float*) malloc (64 * sizeof(float)); 
	float *Res_Layer6_Gamma_CPU = (float*) malloc (128 * sizeof(float));
	float *Res_Layer7_Gamma_CPU = (float*) malloc (128 * sizeof(float)); 
	float *Res_Layer8_Gamma_CPU = (float*) malloc (128 * sizeof(float)); 
    float *Res_Layer9_Gamma_CPU = (float*) malloc (128 * sizeof(float)); 
	float *Res_Layer10_Gamma_CPU = (float*) malloc (256 * sizeof(float)); 
	float *Res_Layer11_Gamma_CPU = (float*) malloc (256 * sizeof(float)); 
	float *Res_Layer12_Gamma_CPU = (float*) malloc (256 * sizeof(float)); 
	float *Res_Layer13_Gamma_CPU = (float*) malloc (256 * sizeof(float)); 
	float *Res_Layer14_Gamma_CPU = (float*) malloc (512 * sizeof(float));
	float *Res_Layer15_Gamma_CPU = (float*) malloc (512 * sizeof(float)); 
	float *Res_Layer16_Gamma_CPU = (float*) malloc (512 * sizeof(float)); 
	float *Res_Layer17_Gamma_CPU = (float*) malloc (512 * sizeof(float));
	float *Res_Block3_Gamma_CPU = (float*) malloc (128 * sizeof(float)); 
	float *Res_Block4_Gamma_CPU = (float*) malloc (256 * sizeof(float));
	float *Res_Block5_Gamma_CPU = (float*) malloc (512 * sizeof(float)); 
    
	float *Res_Layer1_Beta_CPU = (float*) malloc (64 * sizeof(float));
	float *Res_Layer2_Beta_CPU = (float*) malloc (64 * sizeof(float)); 
	float *Res_Layer3_Beta_CPU = (float*) malloc (64 * sizeof(float)); 
	float *Res_Layer4_Beta_CPU = (float*) malloc (64 * sizeof(float)); 
	float *Res_Layer5_Beta_CPU = (float*) malloc (64 * sizeof(float)); 
	float *Res_Layer6_Beta_CPU = (float*) malloc (128 * sizeof(float));
	float *Res_Layer7_Beta_CPU = (float*) malloc (128 * sizeof(float)); 
	float *Res_Layer8_Beta_CPU = (float*) malloc (128 * sizeof(float)); 
    float *Res_Layer9_Beta_CPU = (float*) malloc (128 * sizeof(float)); 
	float *Res_Layer10_Beta_CPU = (float*) malloc (256 * sizeof(float)); 
	float *Res_Layer11_Beta_CPU = (float*) malloc (256 * sizeof(float)); 
	float *Res_Layer12_Beta_CPU = (float*) malloc (256 * sizeof(float)); 
	float *Res_Layer13_Beta_CPU = (float*) malloc (256 * sizeof(float)); 
	float *Res_Layer14_Beta_CPU = (float*) malloc (512 * sizeof(float));
	float *Res_Layer15_Beta_CPU = (float*) malloc (512 * sizeof(float)); 
	float *Res_Layer16_Beta_CPU = (float*) malloc (512 * sizeof(float)); 
	float *Res_Layer17_Beta_CPU = (float*) malloc (512 * sizeof(float));
	float *Res_Block3_Beta_CPU = (float*) malloc (128 * sizeof(float));
	float *Res_Block4_Beta_CPU = (float*) malloc (256 * sizeof(float));
	float *Res_Block5_Beta_CPU = (float*) malloc (512 * sizeof(float));
   
	float *Res_mean1_CPU = (float*) malloc (64 * sizeof(float));
	float *Res_mean2_CPU = (float*) malloc (64 * sizeof(float)); 
	float *Res_mean3_CPU = (float*) malloc (64 * sizeof(float)); 
	float *Res_mean4_CPU = (float*) malloc (64 * sizeof(float)); 
	float *Res_mean5_CPU = (float*) malloc (64 * sizeof(float)); 
	float *Res_mean6_CPU = (float*) malloc (128 * sizeof(float));
	float *Res_mean7_CPU = (float*) malloc (128 * sizeof(float)); 
	float *Res_mean8_CPU = (float*) malloc (128 * sizeof(float)); 
    float *Res_mean9_CPU = (float*) malloc (128 * sizeof(float)); 
	float *Res_mean10_CPU = (float*) malloc (256 * sizeof(float)); 
	float *Res_mean11_CPU = (float*) malloc (256 * sizeof(float)); 
	float *Res_mean12_CPU = (float*) malloc (256 * sizeof(float)); 
	float *Res_mean13_CPU = (float*) malloc (256 * sizeof(float)); 
	float *Res_mean14_CPU = (float*) malloc (512 * sizeof(float));
	float *Res_mean15_CPU = (float*) malloc (512 * sizeof(float)); 
	float *Res_mean16_CPU = (float*) malloc (512 * sizeof(float)); 
	float *Res_mean17_CPU = (float*) malloc (512 * sizeof(float));
	float *Res_Block3_mean_CPU = (float*) malloc (128 * sizeof(float));
	float *Res_Block4_mean_CPU = (float*) malloc (256 * sizeof(float));
	float *Res_Block5_mean_CPU = (float*) malloc (512 * sizeof(float));
   
	float *Res_var1_CPU = (float*) malloc (64 * sizeof(float));
	float *Res_var2_CPU = (float*) malloc (64 * sizeof(float)); 
	float *Res_var3_CPU = (float*) malloc (64 * sizeof(float)); 
	float *Res_var4_CPU = (float*) malloc (64 * sizeof(float)); 
	float *Res_var5_CPU = (float*) malloc (64 * sizeof(float)); 
	float *Res_var6_CPU = (float*) malloc (128 * sizeof(float));
	float *Res_var7_CPU = (float*) malloc (128 * sizeof(float)); 
	float *Res_var8_CPU = (float*) malloc (128 * sizeof(float)); 
    float *Res_var9_CPU = (float*) malloc (128 * sizeof(float)); 
	float *Res_var10_CPU = (float*) malloc (256 * sizeof(float)); 
	float *Res_var11_CPU = (float*) malloc (256 * sizeof(float)); 
	float *Res_var12_CPU = (float*) malloc (256 * sizeof(float)); 
	float *Res_var13_CPU = (float*) malloc (256 * sizeof(float)); 
	float *Res_var14_CPU = (float*) malloc (512 * sizeof(float));
	float *Res_var15_CPU = (float*) malloc (512 * sizeof(float)); 
	float *Res_var16_CPU = (float*) malloc (512 * sizeof(float)); 
	float *Res_var17_CPU = (float*) malloc (512 * sizeof(float));
	float *Res_Block3_var_CPU = (float*) malloc (128 * sizeof(float));
	float *Res_Block4_var_CPU = (float*) malloc (256 * sizeof(float));
	float *Res_Block5_var_CPU = (float*) malloc (512 * sizeof(float));
   
	float *Res_FC_bias_CPU = (float*) malloc (1000* sizeof(float));
	float *Res_FC_Weights_CPU = (float*) malloc ((512*1000) * sizeof(float));

	read_parameter("data_resnet18/conv_data/conv1.txt", Res_Layer1_Weights_CPU);
	read_parameter("data_resnet18/conv_data/conv2.txt", Res_Layer2_Weights_CPU);
	read_parameter("data_resnet18/conv_data/conv3.txt", Res_Layer3_Weights_CPU);
	read_parameter("data_resnet18/conv_data/conv4.txt", Res_Layer4_Weights_CPU);
	read_parameter("data_resnet18/conv_data/conv5.txt", Res_Layer5_Weights_CPU);
	read_parameter("data_resnet18/conv_data/conv6.txt", Res_Layer6_Weights_CPU);
	read_parameter("data_resnet18/conv_data/conv7.txt", Res_Layer7_Weights_CPU);
	read_parameter("data_resnet18/conv_data/conv8.txt", Res_Layer8_Weights_CPU);
 	read_parameter("data_resnet18/conv_data/conv9.txt", Res_Layer9_Weights_CPU);
	read_parameter("data_resnet18/conv_data/conv10.txt", Res_Layer10_Weights_CPU);
	read_parameter("data_resnet18/conv_data/conv11.txt", Res_Layer11_Weights_CPU);
	read_parameter("data_resnet18/conv_data/conv12.txt", Res_Layer12_Weights_CPU);
	read_parameter("data_resnet18/conv_data/conv13.txt", Res_Layer13_Weights_CPU);
	read_parameter("data_resnet18/conv_data/conv14.txt", Res_Layer14_Weights_CPU);
	read_parameter("data_resnet18/conv_data/conv15.txt", Res_Layer15_Weights_CPU);
	read_parameter("data_resnet18/conv_data/conv16.txt", Res_Layer16_Weights_CPU);
	read_parameter("data_resnet18/conv_data/conv17.txt", Res_Layer17_Weights_CPU);
	read_parameter("data_resnet18/conv_data/conv_block3.txt", Res_Block3_Weights_CPU);
	read_parameter("data_resnet18/conv_data/conv_block4.txt", Res_Block4_Weights_CPU);
	read_parameter("data_resnet18/conv_data/conv_block5.txt", Res_Block5_Weights_CPU);

	read_parameter("data_resnet18/gamma_data/gamma1.txt", Res_Layer1_Gamma_CPU);
	read_parameter("data_resnet18/gamma_data/gamma2.txt", Res_Layer2_Gamma_CPU);
	read_parameter("data_resnet18/gamma_data/gamma3.txt", Res_Layer3_Gamma_CPU);
	read_parameter("data_resnet18/gamma_data/gamma4.txt", Res_Layer4_Gamma_CPU);
	read_parameter("data_resnet18/gamma_data/gamma5.txt", Res_Layer5_Gamma_CPU);
	read_parameter("data_resnet18/gamma_data/gamma6.txt", Res_Layer6_Gamma_CPU);
	read_parameter("data_resnet18/gamma_data/gamma7.txt", Res_Layer7_Gamma_CPU);
	read_parameter("data_resnet18/gamma_data/gamma8.txt", Res_Layer8_Gamma_CPU);
 	read_parameter("data_resnet18/gamma_data/gamma9.txt", Res_Layer9_Gamma_CPU);
	read_parameter("data_resnet18/gamma_data/gamma10.txt", Res_Layer10_Gamma_CPU);
	read_parameter("data_resnet18/gamma_data/gamma11.txt", Res_Layer11_Gamma_CPU);
	read_parameter("data_resnet18/gamma_data/gamma12.txt", Res_Layer12_Gamma_CPU);
	read_parameter("data_resnet18/gamma_data/gamma13.txt", Res_Layer13_Gamma_CPU);
	read_parameter("data_resnet18/gamma_data/gamma14.txt", Res_Layer14_Gamma_CPU);
	read_parameter("data_resnet18/gamma_data/gamma15.txt", Res_Layer15_Gamma_CPU);
	read_parameter("data_resnet18/gamma_data/gamma16.txt", Res_Layer16_Gamma_CPU);
	read_parameter("data_resnet18/gamma_data/gamma17.txt", Res_Layer17_Gamma_CPU);
	read_parameter("data_resnet18/gamma_data/gamma_block3.txt", Res_Block3_Gamma_CPU);
	read_parameter("data_resnet18/gamma_data/gamma_block4.txt", Res_Block4_Gamma_CPU);
	read_parameter("data_resnet18/gamma_data/gamma_block5.txt", Res_Block5_Gamma_CPU);

	read_parameter("data_resnet18/beta_data/beta1.txt", Res_Layer1_Beta_CPU);
	read_parameter("data_resnet18/beta_data/beta2.txt", Res_Layer2_Beta_CPU);
	read_parameter("data_resnet18/beta_data/beta3.txt", Res_Layer3_Beta_CPU);
	read_parameter("data_resnet18/beta_data/beta4.txt", Res_Layer4_Beta_CPU);
	read_parameter("data_resnet18/beta_data/beta5.txt", Res_Layer5_Beta_CPU);
	read_parameter("data_resnet18/beta_data/beta6.txt", Res_Layer6_Beta_CPU);
	read_parameter("data_resnet18/beta_data/beta7.txt", Res_Layer7_Beta_CPU);
	read_parameter("data_resnet18/beta_data/beta8.txt", Res_Layer8_Beta_CPU);
 	read_parameter("data_resnet18/beta_data/beta9.txt", Res_Layer9_Beta_CPU);
	read_parameter("data_resnet18/beta_data/beta10.txt", Res_Layer10_Beta_CPU);
	read_parameter("data_resnet18/beta_data/beta11.txt", Res_Layer11_Beta_CPU);
	read_parameter("data_resnet18/beta_data/beta12.txt", Res_Layer12_Beta_CPU);
	read_parameter("data_resnet18/beta_data/beta13.txt", Res_Layer13_Beta_CPU);
	read_parameter("data_resnet18/beta_data/beta14.txt", Res_Layer14_Beta_CPU);
	read_parameter("data_resnet18/beta_data/beta15.txt", Res_Layer15_Beta_CPU);
	read_parameter("data_resnet18/beta_data/beta16.txt", Res_Layer16_Beta_CPU);
	read_parameter("data_resnet18/beta_data/beta17.txt", Res_Layer17_Beta_CPU);
	read_parameter("data_resnet18/beta_data/beta_block3.txt", Res_Block3_Beta_CPU);
	read_parameter("data_resnet18/beta_data/beta_block4.txt", Res_Block4_Beta_CPU);
	read_parameter("data_resnet18/beta_data/beta_block5.txt", Res_Block5_Beta_CPU);

	read_parameter("data_resnet18/mean_data/mean1.txt", Res_mean1_CPU);
	read_parameter("data_resnet18/mean_data/mean2.txt", Res_mean2_CPU);
	read_parameter("data_resnet18/mean_data/mean3.txt", Res_mean3_CPU);
	read_parameter("data_resnet18/mean_data/mean4.txt", Res_mean4_CPU);
	read_parameter("data_resnet18/mean_data/mean5.txt", Res_mean5_CPU);
	read_parameter("data_resnet18/mean_data/mean6.txt", Res_mean6_CPU);
	read_parameter("data_resnet18/mean_data/mean7.txt", Res_mean7_CPU);
	read_parameter("data_resnet18/mean_data/mean8.txt", Res_mean8_CPU);
 	read_parameter("data_resnet18/mean_data/mean9.txt", Res_mean9_CPU);
	read_parameter("data_resnet18/mean_data/mean10.txt", Res_mean10_CPU);
	read_parameter("data_resnet18/mean_data/mean11.txt", Res_mean11_CPU);
	read_parameter("data_resnet18/mean_data/mean12.txt", Res_mean12_CPU);
	read_parameter("data_resnet18/mean_data/mean13.txt", Res_mean13_CPU);
	read_parameter("data_resnet18/mean_data/mean14.txt", Res_mean14_CPU);
	read_parameter("data_resnet18/mean_data/mean15.txt", Res_mean15_CPU);
	read_parameter("data_resnet18/mean_data/mean16.txt", Res_mean16_CPU);
	read_parameter("data_resnet18/mean_data/mean17.txt", Res_mean17_CPU);
	read_parameter("data_resnet18/mean_data/mean_block3.txt", Res_Block3_mean_CPU);
	read_parameter("data_resnet18/mean_data/mean_block4.txt", Res_Block4_mean_CPU);
	read_parameter("data_resnet18/mean_data/mean_block5.txt", Res_Block5_mean_CPU);

	read_parameter("data_resnet18/var_data/var1.txt", Res_var1_CPU);
	read_parameter("data_resnet18/var_data/var2.txt", Res_var2_CPU);
	read_parameter("data_resnet18/var_data/var3.txt", Res_var3_CPU);
	read_parameter("data_resnet18/var_data/var4.txt", Res_var4_CPU);
	read_parameter("data_resnet18/var_data/var5.txt", Res_var5_CPU);
	read_parameter("data_resnet18/var_data/var6.txt", Res_var6_CPU);
	read_parameter("data_resnet18/var_data/var7.txt", Res_var7_CPU);
	read_parameter("data_resnet18/var_data/var8.txt", Res_var8_CPU);
 	read_parameter("data_resnet18/var_data/var9.txt", Res_var9_CPU);
	read_parameter("data_resnet18/var_data/var10.txt", Res_var10_CPU);
	read_parameter("data_resnet18/var_data/var11.txt", Res_var11_CPU);
	read_parameter("data_resnet18/var_data/var12.txt", Res_var12_CPU);
	read_parameter("data_resnet18/var_data/var13.txt", Res_var13_CPU);
	read_parameter("data_resnet18/var_data/var14.txt", Res_var14_CPU);
	read_parameter("data_resnet18/var_data/var15.txt", Res_var15_CPU);
	read_parameter("data_resnet18/var_data/var16.txt", Res_var16_CPU);
	read_parameter("data_resnet18/var_data/var17.txt", Res_var17_CPU);
	read_parameter("data_resnet18/var_data/var_block3.txt", Res_Block3_var_CPU);
	read_parameter("data_resnet18/var_data/var_block4.txt", Res_Block4_var_CPU);
	read_parameter("data_resnet18/var_data/var_block5.txt", Res_Block5_var_CPU);

	read_parameter("data_resnet18/fc_data/fc1_bias.txt", Res_FC_bias_CPU);
	read_parameter("data_resnet18/fc_data/fc1_weight.txt", Res_FC_Weights_CPU);

    float *Res_Layer1_Neurons_data;
	float *Res_Layer1_Weights_data, *Res_Layer2_Weights_data, *Res_Layer3_Weights_data, *Res_Layer4_Weights_data, 
			*Res_Layer5_Weights_data, *Res_Layer6_Weights_data, *Res_Layer7_Weights_data, *Res_Layer8_Weights_data, 
			*Res_Layer9_Weights_data, *Res_Layer10_Weights_data, *Res_Layer11_Weights_data, *Res_Layer12_Weights_data, 
			*Res_Layer13_Weights_data, *Res_Layer14_Weights_data, *Res_Layer15_Weights_data, *Res_Layer16_Weights_data, 
			*Res_Layer17_Weights_data, *Res_Block3_Weights_data, *Res_Block4_Weights_data, *Res_Block5_Weights_data; 
	float *Res_Layer1_Gamma_data, *Res_Layer2_Gamma_data, *Res_Layer3_Gamma_data, *Res_Layer4_Gamma_data,
			*Res_Layer5_Gamma_data, *Res_Layer6_Gamma_data, *Res_Layer7_Gamma_data, *Res_Layer8_Gamma_data,
			*Res_Layer9_Gamma_data, *Res_Layer10_Gamma_data, *Res_Layer11_Gamma_data, *Res_Layer12_Gamma_data,
			*Res_Layer13_Gamma_data, *Res_Layer14_Gamma_data, *Res_Layer15_Gamma_data, *Res_Layer16_Gamma_data,
			*Res_Layer17_Gamma_data, *Res_Block3_Gamma_data, *Res_Block4_Gamma_data, *Res_Block5_Gamma_data;
	float *Res_Layer1_Beta_data, *Res_Layer2_Beta_data, *Res_Layer3_Beta_data, *Res_Layer4_Beta_data,
			*Res_Layer5_Beta_data, *Res_Layer6_Beta_data, *Res_Layer7_Beta_data, *Res_Layer8_Beta_data,
			*Res_Layer9_Beta_data, *Res_Layer10_Beta_data, *Res_Layer11_Beta_data, *Res_Layer12_Beta_data,
			*Res_Layer13_Beta_data, *Res_Layer14_Beta_data, *Res_Layer15_Beta_data, *Res_Layer16_Beta_data,
			*Res_Layer17_Beta_data, *Res_Block3_Beta_data, *Res_Block4_Beta_data, *Res_Block5_Beta_data;
	float *Res_mean1_data, *Res_mean2_data, *Res_mean3_data, *Res_mean4_data, *Res_mean5_data,
			*Res_mean6_data, *Res_mean7_data, *Res_mean8_data, *Res_mean9_data, *Res_mean10_data,
			*Res_mean11_data, *Res_mean12_data, *Res_mean13_data, *Res_mean14_data, *Res_mean15_data,
			*Res_mean16_data, *Res_mean17_data, *Res_Block3_mean_data, *Res_Block4_mean_data, *Res_Block5_mean_data;
	float *Res_var1_data, *Res_var2_data, *Res_var3_data, *Res_var4_data, *Res_var5_data,
			*Res_var6_data, *Res_var7_data, *Res_var8_data, *Res_var9_data, *Res_var10_data,
			*Res_var11_data, *Res_var12_data, *Res_var13_data, *Res_var14_data, *Res_var15_data,
			*Res_var16_data, *Res_var17_data, *Res_Block3_var_data, *Res_Block4_var_data, *Res_Block5_var_data;
	float *Res_FC_bias_data, *Res_FC_Weights_data; 

	hipMalloc((void**) &Res_Layer1_Neurons_data, INPUT_SIZE * sizeof(float)); //224*224*3
	hipMalloc((void**) &Res_Layer1_Weights_data, sizeof(float) * (7*7*3*64));
	hipMalloc((void**) &Res_Layer1_Gamma_data, sizeof(float) * 64);
	hipMalloc((void**) &Res_Layer1_Beta_data, sizeof(float) * 64);
	hipMalloc((void**) &Res_mean1_data, sizeof(float) * 64);
	hipMalloc((void**) &Res_var1_data, sizeof(float) * 64);
	hipMalloc((void**) &Res_Layer2_Weights_data, sizeof(float) * (3*3*64*64));
	hipMalloc((void**) &Res_Layer2_Gamma_data, sizeof(float) * 64);
	hipMalloc((void**) &Res_Layer2_Beta_data, sizeof(float) * 64);
	hipMalloc((void**) &Res_mean2_data, sizeof(float) * 64);
	hipMalloc((void**) &Res_var2_data, sizeof(float) * 64);
	hipMalloc((void**) &Res_Layer3_Weights_data, sizeof(float) * (3*3*64*64));
	hipMalloc((void**) &Res_Layer3_Gamma_data, sizeof(float) * 64);
	hipMalloc((void**) &Res_Layer3_Beta_data, sizeof(float) * 64);
	hipMalloc((void**) &Res_mean3_data, sizeof(float) * 64);
	hipMalloc((void**) &Res_var3_data, sizeof(float) * 64);
	hipMalloc((void**) &Res_Layer4_Weights_data, sizeof(float) * (3*3*64*64));
	hipMalloc((void**) &Res_Layer4_Gamma_data, sizeof(float) * 64);
	hipMalloc((void**) &Res_Layer4_Beta_data, sizeof(float) * 64);
	hipMalloc((void**) &Res_mean4_data, sizeof(float) * 64);
	hipMalloc((void**) &Res_var4_data, sizeof(float) * 64);
	hipMalloc((void**) &Res_Layer5_Weights_data, sizeof(float) * (3*3*64*64));
	hipMalloc((void**) &Res_Layer5_Gamma_data, sizeof(float) * 64);
	hipMalloc((void**) &Res_Layer5_Beta_data, sizeof(float) * 64);
	hipMalloc((void**) &Res_mean5_data, sizeof(float) * 64);
	hipMalloc((void**) &Res_var5_data, sizeof(float) * 64);
	hipMalloc((void**) &Res_Layer6_Weights_data, sizeof(float) * (3*3*64*128));
	hipMalloc((void**) &Res_Layer6_Gamma_data, sizeof(float) * 128);
	hipMalloc((void**) &Res_Layer6_Beta_data, sizeof(float) * 128);
	hipMalloc((void**) &Res_mean6_data, sizeof(float) * 128);
	hipMalloc((void**) &Res_var6_data, sizeof(float) * 128);
	hipMalloc((void**) &Res_Layer7_Weights_data, sizeof(float) * (3*3*128*128));
	hipMalloc((void**) &Res_Layer7_Gamma_data, sizeof(float) * 128);
	hipMalloc((void**) &Res_Layer7_Beta_data, sizeof(float) * 128);
	hipMalloc((void**) &Res_mean7_data, sizeof(float) * 128);
	hipMalloc((void**) &Res_var7_data, sizeof(float) * 128);
	hipMalloc((void**) &Res_Block3_Weights_data, sizeof(float) * (1*1*64*128));
	hipMalloc((void**) &Res_Block3_Gamma_data, sizeof(float) * 128);
	hipMalloc((void**) &Res_Block3_Beta_data, sizeof(float) * 128);
	hipMalloc((void**) &Res_Block3_mean_data, sizeof(float) * 128);
	hipMalloc((void**) &Res_Block3_var_data, sizeof(float) * 128);
	hipMalloc((void**) &Res_Layer8_Weights_data, sizeof(float) * (3*3*128*128));
	hipMalloc((void**) &Res_Layer8_Gamma_data, sizeof(float) * 128);
	hipMalloc((void**) &Res_Layer8_Beta_data, sizeof(float) * 128);
	hipMalloc((void**) &Res_mean8_data, sizeof(float) * 128);
	hipMalloc((void**) &Res_var8_data, sizeof(float) * 128);
	hipMalloc((void**) &Res_Layer9_Weights_data, sizeof(float) * (3*3*128*128));
	hipMalloc((void**) &Res_Layer9_Gamma_data, sizeof(float) * 128);
	hipMalloc((void**) &Res_Layer9_Beta_data, sizeof(float) * 128);
	hipMalloc((void**) &Res_mean9_data, sizeof(float) * 128);
	hipMalloc((void**) &Res_var9_data, sizeof(float) * 128);
	hipMalloc((void**) &Res_Layer10_Weights_data, sizeof(float) * (3*3*128*256));
	hipMalloc((void**) &Res_Layer10_Gamma_data, sizeof(float) * 256);
	hipMalloc((void**) &Res_Layer10_Beta_data, sizeof(float) * 256);
	hipMalloc((void**) &Res_mean10_data, sizeof(float) * 256);
	hipMalloc((void**) &Res_var10_data, sizeof(float) * 256);
	hipMalloc((void**) &Res_Layer11_Weights_data, sizeof(float) * (3*3*256*256));	
	hipMalloc((void**) &Res_Layer11_Gamma_data, sizeof(float) * 256);
	hipMalloc((void**) &Res_Layer11_Beta_data, sizeof(float) * 256);
	hipMalloc((void**) &Res_mean11_data, sizeof(float) * 256);
	hipMalloc((void**) &Res_var11_data, sizeof(float) * 256);
	hipMalloc((void**) &Res_Block4_Weights_data, sizeof(float) * (1*1*128*256));
	hipMalloc((void**) &Res_Block4_Gamma_data, sizeof(float) * 256);
	hipMalloc((void**) &Res_Block4_Beta_data, sizeof(float) * 256);
	hipMalloc((void**) &Res_Block4_mean_data, sizeof(float) * 256);
	hipMalloc((void**) &Res_Block4_var_data, sizeof(float) * 256);
	hipMalloc((void**) &Res_Layer12_Weights_data, sizeof(float) * (3*3*256*256));
	hipMalloc((void**) &Res_Layer12_Gamma_data, sizeof(float) * 256);
	hipMalloc((void**) &Res_Layer12_Beta_data, sizeof(float) * 256);
	hipMalloc((void**) &Res_mean12_data, sizeof(float) * 256);
	hipMalloc((void**) &Res_var12_data, sizeof(float) * 256);
	hipMalloc((void**) &Res_Layer13_Weights_data, sizeof(float) * (3*3*256*256));
	hipMalloc((void**) &Res_Layer13_Gamma_data, sizeof(float) * 256);
	hipMalloc((void**) &Res_Layer13_Beta_data, sizeof(float) * 256);
	hipMalloc((void**) &Res_mean13_data, sizeof(float) * 256);
	hipMalloc((void**) &Res_var13_data, sizeof(float) * 256);
	hipMalloc((void**) &Res_Layer14_Weights_data, sizeof(float) * (3*3*256*512));
	hipMalloc((void**) &Res_Layer14_Gamma_data, sizeof(float) * 512);
	hipMalloc((void**) &Res_Layer14_Beta_data, sizeof(float) * 512);
	hipMalloc((void**) &Res_mean14_data, sizeof(float) * 512);
	hipMalloc((void**) &Res_var14_data, sizeof(float) * 512);
	hipMalloc((void**) &Res_Layer15_Weights_data, sizeof(float) * (3*3*512*512));
	hipMalloc((void**) &Res_Layer15_Gamma_data, sizeof(float) * 512);
	hipMalloc((void**) &Res_Layer15_Beta_data, sizeof(float) * 512);
	hipMalloc((void**) &Res_mean15_data, sizeof(float) * 512);
	hipMalloc((void**) &Res_var15_data, sizeof(float) * 512);
	hipMalloc((void**) &Res_Block5_Weights_data, sizeof(float) * (1*1*256*512));
	hipMalloc((void**) &Res_Block5_Gamma_data, sizeof(float) * 521);
	hipMalloc((void**) &Res_Block5_Beta_data, sizeof(float) * 512);
	hipMalloc((void**) &Res_Block5_mean_data, sizeof(float) * 512);
	hipMalloc((void**) &Res_Block5_var_data, sizeof(float) * 512);
	hipMalloc((void**) &Res_Layer16_Weights_data, sizeof(float) * (3*3*512*512));
	hipMalloc((void**) &Res_Layer16_Gamma_data, sizeof(float) * 512);
	hipMalloc((void**) &Res_Layer16_Beta_data, sizeof(float) * 512);	
	hipMalloc((void**) &Res_mean16_data, sizeof(float) * 512);
	hipMalloc((void**) &Res_var16_data, sizeof(float) * 512);
	hipMalloc((void**) &Res_Layer17_Weights_data, sizeof(float) * (3*3*512*512));
	hipMalloc((void**) &Res_Layer17_Gamma_data, sizeof(float) * 512);
	hipMalloc((void**) &Res_Layer17_Beta_data, sizeof(float) * 512);
	hipMalloc((void**) &Res_mean17_data, sizeof(float) * 512);
	hipMalloc((void**) &Res_var17_data, sizeof(float) * 512);
	hipMalloc((void**) &Res_FC_bias_data, sizeof(float) * 1000);
	hipMalloc((void**) &Res_FC_Weights_data, sizeof(float) * (512*1000));

	hipMemcpy(Res_Layer1_Neurons_data, Res_Layer1_Neurons_CPU, INPUT_SIZE * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(Res_Layer1_Weights_data, Res_Layer1_Weights_CPU, sizeof(float) * (7*7*3*64), hipMemcpyHostToDevice);
	hipMemcpy(Res_Layer1_Gamma_data, Res_Layer1_Gamma_CPU, sizeof(float) * 64, hipMemcpyHostToDevice);
	hipMemcpy(Res_Layer1_Beta_data, Res_Layer1_Beta_CPU, sizeof(float) * 64, hipMemcpyHostToDevice);
	hipMemcpy(Res_mean1_data, Res_mean1_CPU, sizeof(float) * 64, hipMemcpyHostToDevice);
	hipMemcpy(Res_var1_data, Res_var1_CPU, sizeof(float) * 64, hipMemcpyHostToDevice);
	hipMemcpy(Res_Layer2_Weights_data, Res_Layer2_Weights_CPU, sizeof(float) * (3*3*64*64), hipMemcpyHostToDevice);
	hipMemcpy(Res_Layer2_Gamma_data, Res_Layer2_Gamma_CPU, sizeof(float) * 64, hipMemcpyHostToDevice);
	hipMemcpy(Res_Layer2_Beta_data, Res_Layer2_Beta_CPU, sizeof(float) * 64, hipMemcpyHostToDevice);
	hipMemcpy(Res_mean2_data, Res_mean2_CPU, sizeof(float) * 64, hipMemcpyHostToDevice);
	hipMemcpy(Res_var2_data, Res_var2_CPU, sizeof(float) * 64, hipMemcpyHostToDevice);
	hipMemcpy(Res_Layer3_Weights_data, Res_Layer3_Weights_CPU, sizeof(float) * (3*3*64*64), hipMemcpyHostToDevice);
	hipMemcpy(Res_Layer3_Gamma_data, Res_Layer3_Gamma_CPU, sizeof(float) * 64, hipMemcpyHostToDevice);
	hipMemcpy(Res_Layer3_Beta_data, Res_Layer3_Beta_CPU, sizeof(float) * 64, hipMemcpyHostToDevice);
	hipMemcpy(Res_mean3_data, Res_mean3_CPU, sizeof(float) * 64, hipMemcpyHostToDevice);
	hipMemcpy(Res_var3_data, Res_var3_CPU, sizeof(float) * 64, hipMemcpyHostToDevice);
	hipMemcpy(Res_Layer4_Weights_data, Res_Layer4_Weights_CPU, sizeof(float) * (3*3*64*64), hipMemcpyHostToDevice);	
	hipMemcpy(Res_Layer4_Gamma_data, Res_Layer4_Gamma_CPU, sizeof(float) * 64, hipMemcpyHostToDevice);
	hipMemcpy(Res_Layer4_Beta_data, Res_Layer4_Beta_CPU, sizeof(float) * 64, hipMemcpyHostToDevice);
	hipMemcpy(Res_mean4_data, Res_mean4_CPU, sizeof(float) * 64, hipMemcpyHostToDevice);
	hipMemcpy(Res_var4_data, Res_var4_CPU, sizeof(float) * 64, hipMemcpyHostToDevice);
	hipMemcpy(Res_Layer5_Weights_data, Res_Layer5_Weights_CPU, sizeof(float) * (3*3*64*64), hipMemcpyHostToDevice);
	hipMemcpy(Res_Layer5_Gamma_data, Res_Layer5_Gamma_CPU, sizeof(float) * 64, hipMemcpyHostToDevice);
	hipMemcpy(Res_Layer5_Beta_data, Res_Layer5_Beta_CPU, sizeof(float) * 64, hipMemcpyHostToDevice);
	hipMemcpy(Res_mean5_data, Res_mean5_CPU, sizeof(float) * 64, hipMemcpyHostToDevice);
	hipMemcpy(Res_var5_data, Res_var5_CPU, sizeof(float) * 64, hipMemcpyHostToDevice);
	hipMemcpy(Res_Layer6_Weights_data, Res_Layer6_Weights_CPU, sizeof(float) * (3*3*64*128), hipMemcpyHostToDevice);	
	hipMemcpy(Res_Layer6_Gamma_data, Res_Layer6_Gamma_CPU, sizeof(float) * 128, hipMemcpyHostToDevice);
	hipMemcpy(Res_Layer6_Beta_data, Res_Layer6_Beta_CPU, sizeof(float) * 128, hipMemcpyHostToDevice);
	hipMemcpy(Res_mean6_data, Res_mean6_CPU, sizeof(float) * 128, hipMemcpyHostToDevice);
	hipMemcpy(Res_var6_data, Res_var6_CPU, sizeof(float) * 128, hipMemcpyHostToDevice);	
	hipMemcpy(Res_Layer7_Weights_data, Res_Layer7_Weights_CPU, sizeof(float) * (3*3*128*128), hipMemcpyHostToDevice);	
	hipMemcpy(Res_Layer7_Gamma_data, Res_Layer7_Gamma_CPU, sizeof(float) * 128, hipMemcpyHostToDevice);
	hipMemcpy(Res_Layer7_Beta_data, Res_Layer7_Beta_CPU, sizeof(float) * 128, hipMemcpyHostToDevice);
	hipMemcpy(Res_mean7_data, Res_mean7_CPU, sizeof(float) * 128, hipMemcpyHostToDevice);
	hipMemcpy(Res_var7_data, Res_var7_CPU, sizeof(float) * 128, hipMemcpyHostToDevice);
	hipMemcpy(Res_Block3_Weights_data, Res_Block3_Weights_CPU, sizeof(float) * (1*1*64*128), hipMemcpyHostToDevice);
	hipMemcpy(Res_Block3_Gamma_data, Res_Block3_Gamma_CPU, sizeof(float) * 128, hipMemcpyHostToDevice);
	hipMemcpy(Res_Block3_Beta_data, Res_Block3_Beta_CPU, sizeof(float) * 128, hipMemcpyHostToDevice);
	hipMemcpy(Res_Block3_mean_data, Res_Block3_mean_CPU, sizeof(float) * 128, hipMemcpyHostToDevice);
	hipMemcpy(Res_Block3_var_data, Res_Block3_var_CPU, sizeof(float) * 128, hipMemcpyHostToDevice);
	hipMemcpy(Res_Layer8_Weights_data, Res_Layer8_Weights_CPU, sizeof(float) * (3*3*128*128), hipMemcpyHostToDevice);
	hipMemcpy(Res_Layer8_Gamma_data, Res_Layer8_Gamma_CPU, sizeof(float) * 128, hipMemcpyHostToDevice);
	hipMemcpy(Res_Layer8_Beta_data, Res_Layer8_Beta_CPU, sizeof(float) * 128, hipMemcpyHostToDevice);	
	hipMemcpy(Res_mean8_data, Res_mean8_CPU, sizeof(float) * 128, hipMemcpyHostToDevice);
	hipMemcpy(Res_var8_data, Res_var8_CPU, sizeof(float) * 128, hipMemcpyHostToDevice);
	hipMemcpy(Res_Layer9_Weights_data, Res_Layer9_Weights_CPU, sizeof(float) * (3*3*128*128), hipMemcpyHostToDevice);	
	hipMemcpy(Res_Layer9_Gamma_data, Res_Layer9_Gamma_CPU, sizeof(float) * 128, hipMemcpyHostToDevice);
	hipMemcpy(Res_Layer9_Beta_data, Res_Layer9_Beta_CPU, sizeof(float) * 128, hipMemcpyHostToDevice);
	hipMemcpy(Res_mean9_data, Res_mean9_CPU, sizeof(float) * 128, hipMemcpyHostToDevice);
	hipMemcpy(Res_var9_data, Res_var9_CPU, sizeof(float) * 128, hipMemcpyHostToDevice);
	hipMemcpy(Res_Layer10_Weights_data, Res_Layer10_Weights_CPU, sizeof(float) * (3*3*128*256), hipMemcpyHostToDevice);
	hipMemcpy(Res_Layer10_Gamma_data, Res_Layer10_Gamma_CPU, sizeof(float) * 256, hipMemcpyHostToDevice);
	hipMemcpy(Res_Layer10_Beta_data, Res_Layer10_Beta_CPU, sizeof(float) * 256, hipMemcpyHostToDevice);
	hipMemcpy(Res_mean10_data, Res_mean10_CPU, sizeof(float) * 256, hipMemcpyHostToDevice);
	hipMemcpy(Res_var10_data, Res_var10_CPU, sizeof(float) * 256, hipMemcpyHostToDevice);
	hipMemcpy(Res_Layer11_Weights_data, Res_Layer11_Weights_CPU, sizeof(float) * (3*3*256*256), hipMemcpyHostToDevice);
	hipMemcpy(Res_Layer11_Gamma_data, Res_Layer11_Gamma_CPU, sizeof(float) * 256, hipMemcpyHostToDevice);
	hipMemcpy(Res_Layer11_Beta_data, Res_Layer11_Beta_CPU, sizeof(float) * 256, hipMemcpyHostToDevice);
	hipMemcpy(Res_mean11_data, Res_mean11_CPU, sizeof(float) * 256, hipMemcpyHostToDevice);
	hipMemcpy(Res_var11_data, Res_var11_CPU, sizeof(float) * 256, hipMemcpyHostToDevice);
	hipMemcpy(Res_Block4_Weights_data, Res_Block4_Weights_CPU, sizeof(float) * (1*1*128*256), hipMemcpyHostToDevice);
	hipMemcpy(Res_Block4_Gamma_data, Res_Block4_Gamma_CPU, sizeof(float) * 256, hipMemcpyHostToDevice);
	hipMemcpy(Res_Block4_Beta_data, Res_Block4_Beta_CPU, sizeof(float) * 256, hipMemcpyHostToDevice);
	hipMemcpy(Res_Block4_mean_data, Res_Block4_mean_CPU, sizeof(float) * 256, hipMemcpyHostToDevice);
	hipMemcpy(Res_Block4_var_data, Res_Block4_var_CPU, sizeof(float) * 256, hipMemcpyHostToDevice);
	hipMemcpy(Res_Layer12_Weights_data, Res_Layer12_Weights_CPU, sizeof(float) * (3*3*256*256), hipMemcpyHostToDevice);
	hipMemcpy(Res_Layer12_Gamma_data, Res_Layer12_Gamma_CPU, sizeof(float) * 256, hipMemcpyHostToDevice);
	hipMemcpy(Res_Layer12_Beta_data, Res_Layer12_Beta_CPU, sizeof(float) * 256, hipMemcpyHostToDevice);
	hipMemcpy(Res_mean12_data, Res_mean12_CPU, sizeof(float) * 256, hipMemcpyHostToDevice);
	hipMemcpy(Res_var12_data, Res_var12_CPU, sizeof(float) * 256, hipMemcpyHostToDevice);
	hipMemcpy(Res_Layer13_Weights_data, Res_Layer13_Weights_CPU, sizeof(float) * (3*3*256*256), hipMemcpyHostToDevice);
	hipMemcpy(Res_Layer13_Gamma_data, Res_Layer13_Gamma_CPU, sizeof(float) * 256, hipMemcpyHostToDevice);
	hipMemcpy(Res_Layer13_Beta_data, Res_Layer13_Beta_CPU, sizeof(float) * 256, hipMemcpyHostToDevice);
	hipMemcpy(Res_mean13_data, Res_mean13_CPU, sizeof(float) * 256, hipMemcpyHostToDevice);
	hipMemcpy(Res_var13_data, Res_var13_CPU, sizeof(float) * 256, hipMemcpyHostToDevice);
	hipMemcpy(Res_Layer14_Weights_data, Res_Layer14_Weights_CPU, sizeof(float) * (3*3*256*512), hipMemcpyHostToDevice);
	hipMemcpy(Res_Layer14_Gamma_data, Res_Layer14_Gamma_CPU, sizeof(float) * 512, hipMemcpyHostToDevice);
	hipMemcpy(Res_Layer14_Beta_data, Res_Layer14_Beta_CPU, sizeof(float) * 512, hipMemcpyHostToDevice);
	hipMemcpy(Res_mean14_data, Res_mean14_CPU, sizeof(float) * 512, hipMemcpyHostToDevice);
	hipMemcpy(Res_var14_data, Res_var14_CPU, sizeof(float) * 512, hipMemcpyHostToDevice);
	hipMemcpy(Res_Layer15_Weights_data, Res_Layer15_Weights_CPU, sizeof(float) * (3*3*512*512), hipMemcpyHostToDevice);
	hipMemcpy(Res_Layer15_Gamma_data, Res_Layer15_Gamma_CPU, sizeof(float) * 512, hipMemcpyHostToDevice);
	hipMemcpy(Res_Layer15_Beta_data, Res_Layer15_Beta_CPU, sizeof(float) * 512, hipMemcpyHostToDevice);
	hipMemcpy(Res_mean15_data, Res_mean15_CPU, sizeof(float) * 512, hipMemcpyHostToDevice);
	hipMemcpy(Res_var15_data, Res_var15_CPU, sizeof(float) * 512, hipMemcpyHostToDevice);
	hipMemcpy(Res_Block5_Weights_data, Res_Block5_Weights_CPU, sizeof(float) * (1*1*256*512), hipMemcpyHostToDevice);
	hipMemcpy(Res_Block5_Gamma_data, Res_Block5_Gamma_CPU, sizeof(float) * 512, hipMemcpyHostToDevice);
	hipMemcpy(Res_Block5_Beta_data, Res_Block5_Beta_CPU, sizeof(float) * 512, hipMemcpyHostToDevice);
	hipMemcpy(Res_Block5_mean_data, Res_Block5_mean_CPU, sizeof(float) * 512, hipMemcpyHostToDevice);
	hipMemcpy(Res_Block5_var_data, Res_Block5_var_CPU, sizeof(float) * 512, hipMemcpyHostToDevice);
	hipMemcpy(Res_Layer16_Weights_data, Res_Layer16_Weights_CPU, sizeof(float) * (3*3*512*512), hipMemcpyHostToDevice);
	hipMemcpy(Res_Layer16_Gamma_data, Res_Layer16_Gamma_CPU, sizeof(float) * 512, hipMemcpyHostToDevice);
	hipMemcpy(Res_Layer16_Beta_data, Res_Layer16_Beta_CPU, sizeof(float) * 512, hipMemcpyHostToDevice);
	hipMemcpy(Res_mean16_data, Res_mean16_CPU, sizeof(float) * 512, hipMemcpyHostToDevice);
	hipMemcpy(Res_var16_data, Res_var16_CPU, sizeof(float) * 512, hipMemcpyHostToDevice);
	hipMemcpy(Res_Layer17_Weights_data, Res_Layer17_Weights_CPU, sizeof(float) * (3*3*512*512), hipMemcpyHostToDevice);
	hipMemcpy(Res_Layer17_Gamma_data, Res_Layer17_Gamma_CPU, sizeof(float) * 512, hipMemcpyHostToDevice);
	hipMemcpy(Res_Layer17_Beta_data, Res_Layer17_Beta_CPU, sizeof(float) * 512, hipMemcpyHostToDevice);
	hipMemcpy(Res_mean17_data, Res_mean17_CPU, sizeof(float) * 512, hipMemcpyHostToDevice);
	hipMemcpy(Res_var17_data, Res_var17_CPU, sizeof(float) * 512, hipMemcpyHostToDevice);
	hipMemcpy(Res_FC_bias_data, Res_FC_bias_CPU, sizeof(float) * 1000, hipMemcpyHostToDevice);
	hipMemcpy(Res_FC_Weights_data, Res_FC_Weights_CPU, sizeof(float) * (512*1000), hipMemcpyHostToDevice);

	*Res_Layer1_Neurons = Res_Layer1_Neurons_data;

	*Res_Layer1_Weights = Res_Layer1_Weights_data;
	*Res_Layer2_Weights = Res_Layer2_Weights_data;
	*Res_Layer3_Weights = Res_Layer3_Weights_data;
	*Res_Layer4_Weights = Res_Layer4_Weights_data;
	*Res_Layer5_Weights = Res_Layer5_Weights_data;
	*Res_Layer6_Weights = Res_Layer6_Weights_data;
	*Res_Layer7_Weights = Res_Layer7_Weights_data;
	*Res_Layer8_Weights = Res_Layer8_Weights_data;
	*Res_Layer9_Weights = Res_Layer9_Weights_data;
	*Res_Layer10_Weights = Res_Layer10_Weights_data;
	*Res_Layer11_Weights = Res_Layer11_Weights_data;
	*Res_Layer12_Weights = Res_Layer12_Weights_data;
	*Res_Layer13_Weights = Res_Layer13_Weights_data;
	*Res_Layer14_Weights = Res_Layer14_Weights_data;
	*Res_Layer15_Weights = Res_Layer15_Weights_data;
	*Res_Layer16_Weights = Res_Layer16_Weights_data;
	*Res_Layer17_Weights = Res_Layer17_Weights_data;
	*Res_Block3_Weights = Res_Block3_Weights_data;
	*Res_Block4_Weights = Res_Block4_Weights_data;
	*Res_Block5_Weights = Res_Block5_Weights_data;
	
	*Res_Layer1_Gamma = Res_Layer1_Gamma_data;
	*Res_Layer2_Gamma = Res_Layer2_Gamma_data;
	*Res_Layer3_Gamma = Res_Layer3_Gamma_data;
	*Res_Layer4_Gamma = Res_Layer4_Gamma_data;
	*Res_Layer5_Gamma = Res_Layer5_Gamma_data;
	*Res_Layer6_Gamma = Res_Layer6_Gamma_data;
	*Res_Layer7_Gamma = Res_Layer7_Gamma_data;
	*Res_Layer8_Gamma = Res_Layer8_Gamma_data;
	*Res_Layer9_Gamma = Res_Layer9_Gamma_data;
	*Res_Layer10_Gamma = Res_Layer10_Gamma_data;
	*Res_Layer11_Gamma = Res_Layer11_Gamma_data;
	*Res_Layer12_Gamma = Res_Layer12_Gamma_data;
	*Res_Layer13_Gamma = Res_Layer13_Gamma_data;
	*Res_Layer14_Gamma = Res_Layer14_Gamma_data;
	*Res_Layer15_Gamma = Res_Layer15_Gamma_data;
	*Res_Layer16_Gamma = Res_Layer16_Gamma_data;
	*Res_Layer17_Gamma = Res_Layer17_Gamma_data;
	*Res_Block3_Gamma = Res_Block3_Gamma_data;
	*Res_Block4_Gamma = Res_Block4_Gamma_data;
	*Res_Block5_Gamma = Res_Block5_Gamma_data;

	*Res_Layer1_Beta = Res_Layer1_Beta_data;
	*Res_Layer2_Beta = Res_Layer2_Beta_data;
	*Res_Layer3_Beta = Res_Layer3_Beta_data;
	*Res_Layer4_Beta = Res_Layer4_Beta_data;
	*Res_Layer5_Beta = Res_Layer5_Beta_data;
	*Res_Layer6_Beta = Res_Layer6_Beta_data;
	*Res_Layer7_Beta = Res_Layer7_Beta_data;
	*Res_Layer8_Beta = Res_Layer8_Beta_data;
	*Res_Layer9_Beta = Res_Layer9_Beta_data;
	*Res_Layer10_Beta = Res_Layer10_Beta_data;
	*Res_Layer11_Beta = Res_Layer11_Beta_data;
	*Res_Layer12_Beta = Res_Layer12_Beta_data;
	*Res_Layer13_Beta = Res_Layer13_Beta_data;
	*Res_Layer14_Beta = Res_Layer14_Beta_data;
	*Res_Layer15_Beta = Res_Layer15_Beta_data;
	*Res_Layer16_Beta = Res_Layer16_Beta_data;
	*Res_Layer17_Beta = Res_Layer17_Beta_data;
	*Res_Block3_Beta = Res_Block3_Beta_data;
	*Res_Block4_Beta = Res_Block4_Beta_data;
	*Res_Block5_Beta = Res_Block5_Beta_data;

	*Res_mean1 = Res_mean1_data;
	*Res_mean2 = Res_mean2_data;
	*Res_mean3 = Res_mean3_data;
	*Res_mean4 = Res_mean4_data;
	*Res_mean5 = Res_mean5_data;
	*Res_mean6 = Res_mean6_data;
	*Res_mean7 = Res_mean7_data;
	*Res_mean8 = Res_mean8_data;
	*Res_mean9 = Res_mean9_data;
	*Res_mean10 = Res_mean10_data;
	*Res_mean11 = Res_mean11_data;
	*Res_mean12 = Res_mean12_data;
	*Res_mean13 = Res_mean13_data;
	*Res_mean14 = Res_mean14_data;
	*Res_mean15 = Res_mean15_data;
	*Res_mean16 = Res_mean16_data;
	*Res_mean17 = Res_mean17_data;
	*Res_Block3_mean = Res_Block3_mean_data;
	*Res_Block4_mean = Res_Block4_mean_data;
	*Res_Block5_mean = Res_Block5_mean_data;

	*Res_var1 = Res_var1_data;
	*Res_var2 = Res_var2_data;
	*Res_var3 = Res_var3_data;
	*Res_var4 = Res_var4_data;
	*Res_var5 = Res_var5_data;
	*Res_var6 = Res_var6_data;
	*Res_var7 = Res_var7_data;
	*Res_var8 = Res_var8_data;
	*Res_var9 = Res_var9_data;
	*Res_var10 = Res_var10_data;
	*Res_var11 = Res_var11_data;
	*Res_var12 = Res_var12_data;
	*Res_var13 = Res_var13_data;
	*Res_var14 = Res_var14_data;
	*Res_var15 = Res_var15_data;
	*Res_var16 = Res_var16_data;
	*Res_var17 = Res_var17_data;
	*Res_Block3_var = Res_Block3_var_data;
	*Res_Block4_var = Res_Block4_var_data;
	*Res_Block5_var = Res_Block5_var_data;

	*Res_FC_bias = Res_FC_bias_data;
	*Res_FC_Weights = Res_FC_Weights_data;

	free(Res_Layer1_Neurons_CPU);

	free(Res_Layer1_Weights_CPU);
    free(Res_Layer2_Weights_CPU);
    free(Res_Layer3_Weights_CPU);
    free(Res_Layer4_Weights_CPU);
    free(Res_Layer5_Weights_CPU);
    free(Res_Layer6_Weights_CPU);
    free(Res_Layer7_Weights_CPU);
    free(Res_Layer8_Weights_CPU);
	free(Res_Layer9_Weights_CPU);
    free(Res_Layer10_Weights_CPU);
    free(Res_Layer11_Weights_CPU);
    free(Res_Layer12_Weights_CPU);
    free(Res_Layer13_Weights_CPU);
    free(Res_Layer14_Weights_CPU);
    free(Res_Layer15_Weights_CPU);
    free(Res_Layer16_Weights_CPU);
	free(Res_Layer17_Weights_CPU);
    free(Res_Block3_Weights_CPU);
    free(Res_Block4_Weights_CPU);
    free(Res_Block5_Weights_CPU);

	free(Res_Layer1_Gamma_CPU);
    free(Res_Layer2_Gamma_CPU);
    free(Res_Layer3_Gamma_CPU);
    free(Res_Layer4_Gamma_CPU);
    free(Res_Layer5_Gamma_CPU);
    free(Res_Layer6_Gamma_CPU);
    free(Res_Layer7_Gamma_CPU);
    free(Res_Layer8_Gamma_CPU);
	free(Res_Layer9_Gamma_CPU);
    free(Res_Layer10_Gamma_CPU);
    free(Res_Layer11_Gamma_CPU);
    free(Res_Layer12_Gamma_CPU);
    free(Res_Layer13_Gamma_CPU);
    free(Res_Layer14_Gamma_CPU);
    free(Res_Layer15_Gamma_CPU);
    free(Res_Layer16_Gamma_CPU);
	free(Res_Layer17_Gamma_CPU);
    free(Res_Block3_Gamma_CPU);
    free(Res_Block4_Gamma_CPU);
    free(Res_Block5_Gamma_CPU);

	free(Res_Layer1_Beta_CPU);
    free(Res_Layer2_Beta_CPU);
    free(Res_Layer3_Beta_CPU);
    free(Res_Layer4_Beta_CPU);
    free(Res_Layer5_Beta_CPU);
    free(Res_Layer6_Beta_CPU);
    free(Res_Layer7_Beta_CPU);
    free(Res_Layer8_Beta_CPU);
	free(Res_Layer9_Beta_CPU);
    free(Res_Layer10_Beta_CPU);
    free(Res_Layer11_Beta_CPU);
    free(Res_Layer12_Beta_CPU);
    free(Res_Layer13_Beta_CPU);
    free(Res_Layer14_Beta_CPU);
    free(Res_Layer15_Beta_CPU);
    free(Res_Layer16_Beta_CPU);
	free(Res_Layer17_Beta_CPU);
    free(Res_Block3_Beta_CPU);
    free(Res_Block4_Beta_CPU);
    free(Res_Block5_Beta_CPU);

	free(Res_mean1_CPU);
	free(Res_mean2_CPU);
	free(Res_mean3_CPU);
	free(Res_mean4_CPU);
	free(Res_mean5_CPU);
	free(Res_mean6_CPU);
	free(Res_mean7_CPU);
	free(Res_mean8_CPU);
	free(Res_mean9_CPU);
	free(Res_mean10_CPU);
	free(Res_mean11_CPU);
	free(Res_mean12_CPU);
	free(Res_mean13_CPU);
	free(Res_mean14_CPU);
	free(Res_mean15_CPU);
	free(Res_mean16_CPU);
	free(Res_mean17_CPU);
	free(Res_Block3_mean_CPU);
	free(Res_Block4_mean_CPU);
	free(Res_Block5_mean_CPU);

	free(Res_var1_CPU);
	free(Res_var2_CPU);
	free(Res_var3_CPU);
	free(Res_var4_CPU);
	free(Res_var5_CPU);
	free(Res_var6_CPU);
	free(Res_var7_CPU);
	free(Res_var8_CPU);
	free(Res_var9_CPU);
	free(Res_var10_CPU);
	free(Res_var11_CPU);
	free(Res_var12_CPU);
	free(Res_var13_CPU);
	free(Res_var14_CPU);
	free(Res_var15_CPU);
	free(Res_var16_CPU);
	free(Res_var17_CPU);
	free(Res_Block3_var_CPU);
	free(Res_Block4_var_CPU);
	free(Res_Block5_var_CPU);

	float *Res_Layer1_bn_data; 
	hipMalloc((void**) &Res_Layer1_bn_data, (64*112*112) * sizeof(float)); //64*112*112
	*Res_Layer1_bn = Res_Layer1_bn_data;

	float *Res_Layer1_pool_data;
    hipMalloc((void**) &Res_Layer1_pool_data, (64*112*112) * sizeof(float)); //64*112*112
	*Res_Layer1_pool = Res_Layer1_pool_data;

    float *Res_Layer2_Neurons_data;
    hipMalloc((void**) &Res_Layer2_Neurons_data, (64*56*56) * sizeof(float)); //64*56*56
	*Res_Layer2_Neurons = Res_Layer2_Neurons_data;

    float *Res_Layer2_bn_data;
    hipMalloc((void**) &Res_Layer2_bn_data, (64*56*56) * sizeof(float)); //64*56*56
	*Res_Layer2_bn = Res_Layer2_bn_data;

    float *Res_Layer3_Neurons_data;
	hipMalloc((void**) &Res_Layer3_Neurons_data, (64*56*56) * sizeof(float)); //64*56*56
	*Res_Layer3_Neurons = Res_Layer3_Neurons_data;

    float *Res_Layer3_bn_data;
	hipMalloc((void**) &Res_Layer3_bn_data, (64*56*56) * sizeof(float)); //64*56*56
	*Res_Layer3_bn = Res_Layer3_bn_data;

    float *Res_Layer3_basic_data;
    hipMalloc((void**) &Res_Layer3_basic_data, (64*56*56) * sizeof(float)); //64*56*56
	*Res_Layer3_basic = Res_Layer3_basic_data;

    float *Res_Layer4_Neurons_data;
    hipMalloc((void**) &Res_Layer4_Neurons_data, (64*56*56) * sizeof(float)); //64*56*56
	*Res_Layer4_Neurons = Res_Layer4_Neurons_data;

    float *Res_Layer4_bn_data;
    hipMalloc((void**) &Res_Layer4_bn_data, (64*56*56) * sizeof(float)); //64*56*56
	*Res_Layer4_bn = Res_Layer4_bn_data;
	
    float *Res_Layer5_Neurons_data;
    hipMalloc((void**) &Res_Layer5_Neurons_data, (64*56*56) * sizeof(float)); //64*56*56
	*Res_Layer5_Neurons = Res_Layer5_Neurons_data;

    float *Res_Layer5_bn_data;
    hipMalloc((void**) &Res_Layer5_bn_data, (64*56*56) * sizeof(float)); //64*56*56
	*Res_Layer5_bn = Res_Layer5_bn_data;

    float *Res_Layer5_basic_data;
    hipMalloc((void**) &Res_Layer5_basic_data, (64*56*56) * sizeof(float)); //64*56*56
	*Res_Layer5_basic = Res_Layer5_basic_data;

    float *Res_Layer6_Neurons_data;
    hipMalloc((void**) &Res_Layer6_Neurons_data, (64*56*56) * sizeof(float)); //64*56*56
	*Res_Layer6_Neurons = Res_Layer6_Neurons_data;

    float *Res_Layer6_bn_data;
    hipMalloc((void**) &Res_Layer6_bn_data, sizeof(float) * (128*28*28)); //128*28*28
	*Res_Layer6_bn = Res_Layer6_bn_data;

    float *Res_Layer7_Neurons_data;
    hipMalloc((void**) &Res_Layer7_Neurons_data, (128*28*28) * sizeof(float)); //128*28*28
	*Res_Layer7_Neurons = Res_Layer7_Neurons_data;

    float *Res_Layer7_bn_data;
    hipMalloc((void**) &Res_Layer7_bn_data, (128*28*28) * sizeof(float)); //128*28*28
	*Res_Layer7_bn = Res_Layer7_bn_data;

    float *Res_Layer7_basic_data;
    hipMalloc((void**) &Res_Layer7_basic_data, (128*28*28) * sizeof(float)); //128*28*28
	*Res_Layer7_basic = Res_Layer7_basic_data;

    float *Res_Block3_bn_data, *Res_Block3_basic_data, *Res_Layer8_Neurons_data;
	hipMalloc((void**) &Res_Block3_bn_data, (128*28*28) * sizeof(float)); //128*28*28
	hipMalloc((void**) &Res_Block3_basic_data, (128*28*28) * sizeof(float)); //128*28*28
	hipMalloc((void**) &Res_Layer8_Neurons_data, (128*28*28) * sizeof(float)); //128*28*28
	*Res_Block3_bn = Res_Block3_bn_data;
	*Res_Block3_basic = Res_Block3_basic_data;
	*Res_Layer8_Neurons = Res_Layer8_Neurons_data;

    float *Res_Layer8_bn_data;
    hipMalloc((void**) &Res_Layer8_bn_data,(128*28*28) * sizeof(float)); //128*28*28
	*Res_Layer8_bn = Res_Layer8_bn_data;

    float *Res_Layer9_Neurons_data;
    hipMalloc((void**) &Res_Layer9_Neurons_data, (128*28*28) * sizeof(float)); //128*28*28
	*Res_Layer9_Neurons = Res_Layer9_Neurons_data;

    float *Res_Layer9_bn_data;
    hipMalloc((void**) &Res_Layer9_bn_data, (128*28*28) * sizeof(float)); //128*28*28
	*Res_Layer9_bn = Res_Layer9_bn_data;

    float *Res_Layer9_basic_data;
    hipMalloc((void**) &Res_Layer9_basic_data, (128*28*28) * sizeof(float)); //128*28*28
	*Res_Layer9_basic = Res_Layer9_basic_data;

    float *Res_Layer10_Neurons_data;
	hipMalloc((void**) &Res_Layer10_Neurons_data, (128*28*28) * sizeof(float)); //128*28*28
	*Res_Layer10_Neurons = Res_Layer10_Neurons_data;

    float *Res_Layer10_bn_data;
    hipMalloc((void**) &Res_Layer10_bn_data, (256*14*14) * sizeof(float)); //256*14*14
	*Res_Layer10_bn = Res_Layer10_bn_data;

    float *Res_Layer11_Neurons_data;
    hipMalloc((void**) &Res_Layer11_Neurons_data, (256*14*14) * sizeof(float)); //256*14*14
	*Res_Layer11_Neurons = Res_Layer11_Neurons_data;

    float *Res_Layer11_bn_data;
    hipMalloc((void**) &Res_Layer11_bn_data, (256*14*14) * sizeof(float)); //256*14*14
	*Res_Layer11_bn = Res_Layer11_bn_data;

    float *Res_Layer11_basic_data;
    hipMalloc((void**) &Res_Layer11_basic_data, (256*14*14) * sizeof(float)); //256*14*14
	*Res_Layer11_basic = Res_Layer11_basic_data;

	float *Res_Block4_bn_data, *Res_Block4_basic_data, *Res_Layer12_Neurons_data;
	hipMalloc((void**) &Res_Block4_bn_data, (256*14*14) * sizeof(float)); //256*14*14
	hipMalloc((void**) &Res_Block4_basic_data, (256*14*14) * sizeof(float)); //256*14*14
	hipMalloc((void**) &Res_Layer12_Neurons_data, (256*14*14) * sizeof(float)); //256*14*14
	*Res_Block4_bn = Res_Block4_bn_data;
	*Res_Block4_basic = Res_Block4_basic_data;
	*Res_Layer12_Neurons = Res_Layer12_Neurons_data;

    float *Res_Layer12_bn_data;
    hipMalloc((void**) &Res_Layer12_bn_data, (256*14*14) * sizeof(float)); //256*14*14
	*Res_Layer12_bn = Res_Layer12_bn_data;

    float *Res_Layer13_Neurons_data;
    hipMalloc((void**) &Res_Layer13_Neurons_data, (256*14*14) * sizeof(float)); //256*14*14
	*Res_Layer13_Neurons = Res_Layer13_Neurons_data;

    float *Res_Layer13_bn_data;
    hipMalloc((void**) &Res_Layer13_bn_data, (256*14*14) * sizeof(float)); //256*14*14
	*Res_Layer13_bn = Res_Layer13_bn_data;

    float *Res_Layer13_basic_data;
    hipMalloc((void**) &Res_Layer13_basic_data, (256*14*14) * sizeof(float)); //256*14*14
	*Res_Layer13_basic = Res_Layer13_basic_data;

    float *Res_Layer14_Neurons_data;
    hipMalloc((void**) &Res_Layer14_Neurons_data, (256*14*14) * sizeof(float)); //256*14*14
	*Res_Layer14_Neurons = Res_Layer14_Neurons_data;

    float *Res_Layer14_bn_data, *Res_Layer15_Neurons_data;
	hipMalloc((void**) &Res_Layer14_bn_data, (512*7*7) * sizeof(float)); //512*7*7
	hipMalloc((void**) &Res_Layer15_Neurons_data, (512*7*7) * sizeof(float)); //512*7*7
	*Res_Layer14_bn = Res_Layer14_bn_data;
	*Res_Layer15_Neurons = Res_Layer15_Neurons_data;

    float *Res_Layer15_bn_data, *Res_Layer15_basic_data;
	hipMalloc((void**) &Res_Layer15_bn_data, (512*7*7) * sizeof(float)); //512*7*7
	hipMalloc((void**) &Res_Layer15_basic_data, (512*7*7) * sizeof(float)); //512*7*7
	*Res_Layer15_bn = Res_Layer15_bn_data;
	*Res_Layer15_basic = Res_Layer15_basic_data;

	float *Res_Block5_bn_data, *Res_Block5_basic_data, *Res_Layer16_Neurons_data;
	hipMalloc((void**) &Res_Block5_bn_data, (512*7*7) * sizeof(float)); //512*7*7
	hipMalloc((void**) &Res_Block5_basic_data, (512*7*7) * sizeof(float)); //512*7*7
	hipMalloc((void**) &Res_Layer16_Neurons_data, (512*7*7) * sizeof(float)); //512*7*7
	*Res_Block5_bn = Res_Block5_bn_data;
	*Res_Block5_basic = Res_Block5_basic_data;
	*Res_Layer16_Neurons = Res_Layer16_Neurons_data;

	float *Res_Layer16_bn_data, *Res_Layer17_Neurons_data;
	hipMalloc((void**) &Res_Layer16_bn_data, (512*7*7) * sizeof(float)); //512*7*7
	hipMalloc((void**) &Res_Layer17_Neurons_data, (512*7*7) * sizeof(float)); //512*7*7
	*Res_Layer16_bn = Res_Layer16_bn_data;
	*Res_Layer17_Neurons = Res_Layer17_Neurons_data;

    float *Res_Layer17_bn_data, *Res_Layer17_basic_data, *Res_Layer18_Neurons_data;
	hipMalloc((void**) &Res_Layer17_bn_data, (512*7*7) * sizeof(float)); //512*7*7
	hipMalloc((void**) &Res_Layer17_basic_data, (512*7*7) * sizeof(float)); //512*7*7
	hipMalloc((void**) &Res_Layer18_Neurons_data, (512*7*7) * sizeof(float)); //512*7*7
	*Res_Layer17_bn = Res_Layer17_bn_data;
	*Res_Layer17_basic = Res_Layer17_basic_data;
	*Res_Layer18_Neurons = Res_Layer18_Neurons_data;

    float *Res_FC_Neurons_data;
	hipMalloc((void**) &Res_FC_Neurons_data, 512 * sizeof(float));
	*Res_FC_Neurons = Res_FC_Neurons_data;

    float *Res_Result_Neurons_data;
    hipMalloc((void**) &Res_Result_Neurons_data, 1000 * sizeof(float)); //1000
	*Res_Result_Neurons = Res_Result_Neurons_data;
}

void res_first_conv(float *Res_Layer1_Neurons,float *Res_Layer1_Weights,float *Res_Layer1_bn,float *Res_Layer1_pool,float *Res_mean1,float *Res_var1,float *Res_Layer1_Gamma,float *Res_Layer1_Beta)
{
    dim3 Block1_Block(64,16,16);
    dim3 Block_Thread(7,7);
	first<<<Block1_Block,Block_Thread>>>(NULL,Res_Layer1_Neurons,Res_Layer1_Weights,Res_Layer1_bn,224,112,2,3,7,3,false,true);
	batchnorm<<<Block1_Block,Block_Thread>>>(Res_Layer1_bn,Res_Layer1_pool,Res_mean1,Res_var1,Res_Layer1_Gamma,Res_Layer1_Beta,112,true);
}

void res_first_pool(float *Res_Layer1_pool,float *Res_Layer2_Neurons)
{
    dim3 Block1_Pool_Block(64,8,8);
    dim3 Block_Thread(7,7);
	max<<<Block1_Pool_Block,Block_Thread>>>(Res_Layer1_pool,Res_Layer2_Neurons,112,56,2,1,3);
}

void res_second_conv(float *Res_Layer2_Neurons,float *Res_Layer2_Weights,float *Res_Layer2_bn,float *Res_Layer3_Neurons,float *Res_mean2,float *Res_var2,float *Res_Layer2_Gamma,float *Res_Layer2_Beta)
{
    dim3 Block2_Block(64,8,8);
    dim3 Block_Thread(7,7);
	conv<<<Block2_Block,Block_Thread>>>(NULL,Res_Layer2_Neurons,Res_Layer2_Weights,Res_Layer2_bn,56,56,1,1,3,64,false,false);
	batchnorm<<<Block2_Block,Block_Thread>>>(Res_Layer2_bn,Res_Layer3_Neurons,Res_mean2,Res_var2,Res_Layer2_Gamma,Res_Layer2_Beta,56,true);
}

void res_third_conv(float *Res_Layer3_Neurons,float *Res_Layer3_Weights,float *Res_Layer3_bn,float *Res_Layer3_basic,float *Res_mean3,float *Res_var3,float *Res_Layer3_Gamma,float *Res_Layer3_Beta)
{
    dim3 Block2_Block(64,8,8);
    dim3 Block_Thread(7,7);
 	conv<<<Block2_Block,Block_Thread>>>(NULL,Res_Layer3_Neurons,Res_Layer3_Weights,Res_Layer3_bn,56,56,1,1,3,64,false,false);
	batchnorm<<<Block2_Block,Block_Thread>>>(Res_Layer3_bn,Res_Layer3_basic,Res_mean3,Res_var3,Res_Layer3_Gamma,Res_Layer3_Beta,56,false);   
}

void res_third_basic(float *Res_Layer2_Neurons,float *Res_Layer3_basic,float *Res_Layer4_Neurons)
{
    dim3 Block2_Block(64,8,8);
    dim3 Block_Thread(7,7);
	basic_block<<<Block2_Block,Block_Thread>>>(Res_Layer2_Neurons,Res_Layer3_basic,Res_Layer4_Neurons,56,true);   
}

void res_fourth_conv(float *Res_Layer4_Neurons,float *Res_Layer4_Weights,float *Res_Layer4_bn,float *Res_Layer5_Neurons,float *Res_mean4,float *Res_var4,float *Res_Layer4_Gamma,float *Res_Layer4_Beta)
{
    dim3 Block2_Block(64,8,8);
    dim3 Block_Thread(7,7);
  	conv<<<Block2_Block,Block_Thread>>>(NULL,Res_Layer4_Neurons,Res_Layer4_Weights,Res_Layer4_bn,56,56,1,1,3,64,false,false);
	batchnorm<<<Block2_Block,Block_Thread>>>(Res_Layer4_bn,Res_Layer5_Neurons,Res_mean4,Res_var4,Res_Layer4_Gamma,Res_Layer4_Beta,56,true);
}

void res_fifth_conv(float *Res_Layer5_Neurons,float *Res_Layer5_Weights,float *Res_Layer5_bn,float *Res_Layer5_basic,float *Res_mean5,float *Res_var5,float *Res_Layer5_Gamma,float *Res_Layer5_Beta)
{
    dim3 Block2_Block(64,8,8);
    dim3 Block_Thread(7,7);    
	conv<<<Block2_Block,Block_Thread>>>(NULL,Res_Layer5_Neurons,Res_Layer5_Weights,Res_Layer5_bn,56,56,1,1,3,64,false,false);
	batchnorm<<<Block2_Block,Block_Thread>>>(Res_Layer5_bn,Res_Layer5_basic,Res_mean5,Res_var5,Res_Layer5_Gamma,Res_Layer5_Beta,56,false);
}

void res_fifth_basic(float *Res_Layer4_Neurons,float *Res_Layer5_basic,float *Res_Layer6_Neurons)
{
    dim3 Block2_Block(64,8,8);
    dim3 Block_Thread(7,7); 
	basic_block<<<Block2_Block,Block_Thread>>>(Res_Layer4_Neurons,Res_Layer5_basic,Res_Layer6_Neurons,56,true);
}

void res_sixth_conv(float *Res_Layer6_Neurons,float *Res_Layer6_Weights,float *Res_Layer6_bn,float *Res_Layer7_Neurons,float *Res_mean6,float *Res_var6,float *Res_Layer6_Gamma,float *Res_Layer6_Beta)
{
    dim3 Block3_Block(128,4,4);
    dim3 Block_Thread(7,7); 
	conv<<<Block3_Block,Block_Thread>>>(NULL,Res_Layer6_Neurons,Res_Layer6_Weights,Res_Layer6_bn,56,28,2,1,3,64,false,false);
	batchnorm<<<Block3_Block,Block_Thread>>>(Res_Layer6_bn,Res_Layer7_Neurons,Res_mean6,Res_var6,Res_Layer6_Gamma,Res_Layer6_Beta,28,true);
}

void res_seventh_conv(float *Res_Layer7_Neurons,float *Res_Layer7_Weights,float *Res_Layer7_bn,float *Res_Layer7_basic,float *Res_mean7,float *Res_var7,float *Res_Layer7_Gamma,float *Res_Layer7_Beta)
{
    dim3 Block3_Block(128,4,4);
    dim3 Block_Thread(7,7); 
	conv<<<Block3_Block,Block_Thread>>>(NULL,Res_Layer7_Neurons,Res_Layer7_Weights,Res_Layer7_bn,28,28,1,1,3,128,false,false);
	batchnorm<<<Block3_Block,Block_Thread>>>(Res_Layer7_bn,Res_Layer7_basic,Res_mean7,Res_var7,Res_Layer7_Gamma,Res_Layer7_Beta,28,false);
}

void res_Block_B_conv(float *Res_Layer6_Neurons,float *Res_Block3_Weights,float *Res_Block3_bn,float *Res_Block3_basic,float *Res_Block3_mean,float *Res_Block3_var,float *Res_Block3_Gamma,float *Res_Block3_Beta)
{
    dim3 Block3_Block(128,4,4);
    dim3 Block_Thread(7,7); 
	conv<<<Block3_Block,Block_Thread>>>(NULL,Res_Layer6_Neurons,Res_Block3_Weights,Res_Block3_bn,56,28,2,0,1,64,false,false); 
	batchnorm<<<Block3_Block,Block_Thread>>>(Res_Block3_bn,Res_Block3_basic,Res_Block3_mean,Res_Block3_var,Res_Block3_Gamma,Res_Block3_Beta,28,false);
}

void res_Block_B_basic(float *Res_Layer7_basic,float *Res_Block3_basic,float *Res_Layer8_Neurons)
{
    dim3 Block3_Block(128,4,4);
    dim3 Block_Thread(7,7); 
	basic_block<<<Block3_Block,Block_Thread>>>(Res_Layer7_basic,Res_Block3_basic,Res_Layer8_Neurons,28,true);
}

void res_eighth_conv(float *Res_Layer8_Neurons,float *Res_Layer8_Weights,float *Res_Layer8_bn,float *Res_Layer9_Neurons,float *Res_mean8,float *Res_var8,float *Res_Layer8_Gamma,float *Res_Layer8_Beta)
{
    dim3 Block3_Block(128,4,4);
    dim3 Block_Thread(7,7); 
	conv<<<Block3_Block,Block_Thread>>>(NULL,Res_Layer8_Neurons,Res_Layer8_Weights,Res_Layer8_bn,28,28,1,1,3,128,false,false);
	batchnorm<<<Block3_Block,Block_Thread>>>(Res_Layer8_bn,Res_Layer9_Neurons,Res_mean8,Res_var8,Res_Layer8_Gamma,Res_Layer8_Beta,28,true);
}

void res_ninth_conv(float *Res_Layer9_Neurons,float *Res_Layer9_Weights,float *Res_Layer9_bn,float *Res_Layer9_basic,float *Res_mean9,float *Res_var9,float *Res_Layer9_Gamma,float *Res_Layer9_Beta)
{
    dim3 Block3_Block(128,4,4);
    dim3 Block_Thread(7,7); 
	conv<<<Block3_Block,Block_Thread>>>(NULL,Res_Layer9_Neurons,Res_Layer9_Weights,Res_Layer9_bn,28,28,1,1,3,128,false,false);
	batchnorm<<<Block3_Block,Block_Thread>>>(Res_Layer9_bn,Res_Layer9_basic,Res_mean9,Res_var9,Res_Layer9_Gamma,Res_Layer9_Beta,28,false);
}

void res_ninth_basic(float *Res_Layer8_Neurons,float *Res_Layer9_basic,float *Res_Layer10_Neurons)
{
    dim3 Block3_Block(128,4,4);
    dim3 Block_Thread(7,7); 
 	basic_block<<<Block3_Block,Block_Thread>>>(Res_Layer8_Neurons,Res_Layer9_basic,Res_Layer10_Neurons,28,true);   
}

void res_tenth_conv(float *Res_Layer10_Neurons,float *Res_Layer10_Weights,float *Res_Layer10_bn,float *Res_Layer11_Neurons,float *Res_mean10,float *Res_var10,float *Res_Layer10_Gamma,float *Res_Layer10_Beta)
{
    dim3 Block4_Block(256,2,2);
    dim3 Block_Thread(7,7); 
	conv<<<Block4_Block,Block_Thread>>>(NULL,Res_Layer10_Neurons,Res_Layer10_Weights,Res_Layer10_bn,28,14,2,1,3,128,false,false);
	batchnorm<<<Block4_Block,Block_Thread>>>(Res_Layer10_bn,Res_Layer11_Neurons,Res_mean10,Res_var10,Res_Layer10_Gamma,Res_Layer10_Beta,14,true);
}

void res_eleventh_conv(float *Res_Layer11_Neurons,float *Res_Layer11_Weights,float *Res_Layer11_bn,float *Res_Layer11_basic,float *Res_mean11,float *Res_var11,float *Res_Layer11_Gamma,float *Res_Layer11_Beta)
{
    dim3 Block4_Block(256,2,2);
    dim3 Block_Thread(7,7);
	conv<<<Block4_Block,Block_Thread>>>(NULL,Res_Layer11_Neurons,Res_Layer11_Weights,Res_Layer11_bn,14,14,1,1,3,256,false,false);
	batchnorm<<<Block4_Block,Block_Thread>>>(Res_Layer11_bn,Res_Layer11_basic,Res_mean11,Res_var11,Res_Layer11_Gamma,Res_Layer11_Beta,14,false);
}

void res_Block_C_conv(float *Res_Layer10_Neurons,float *Res_Block4_Weights,float *Res_Block4_bn,float *Res_Block4_basic,float *Res_Block4_mean,float *Res_Block4_var,float *Res_Block4_Gamma,float *Res_Block4_Beta)
{
    dim3 Block4_Block(256,2,2);
    dim3 Block_Thread(7,7);
	conv<<<Block4_Block,Block_Thread>>>(NULL,Res_Layer10_Neurons,Res_Block4_Weights,Res_Block4_bn,28,14,2,0,1,128,false,false);
	batchnorm<<<Block4_Block,Block_Thread>>>(Res_Block4_bn,Res_Block4_basic,Res_Block4_mean,Res_Block4_var,Res_Block4_Gamma,Res_Block4_Beta,14,false);
}

void res_Block_C_basic(float *Res_Layer11_basic,float *Res_Block4_basic,float *Res_Layer12_Neurons)
{
    dim3 Block4_Block(256,2,2);
    dim3 Block_Thread(7,7);
	basic_block<<<Block4_Block,Block_Thread>>>(Res_Layer11_basic,Res_Block4_basic,Res_Layer12_Neurons,14,true);
}

void res_twelfth_conv(float *Res_Layer12_Neurons,float *Res_Layer12_Weights,float *Res_Layer12_bn,float *Res_Layer13_Neurons,float *Res_mean12,float *Res_var12,float *Res_Layer12_Gamma,float *Res_Layer12_Beta)
{
    dim3 Block4_Block(256,2,2);
    dim3 Block_Thread(7,7);
	conv<<<Block4_Block,Block_Thread>>>(NULL,Res_Layer12_Neurons,Res_Layer12_Weights,Res_Layer12_bn,14,14,1,1,3,256,false,false);
	batchnorm<<<Block4_Block,Block_Thread>>>(Res_Layer12_bn,Res_Layer13_Neurons,Res_mean12,Res_var12,Res_Layer12_Gamma,Res_Layer12_Beta,14,true);
}

void res_thirteenth_conv(float *Res_Layer13_Neurons,float *Res_Layer13_Weights,float *Res_Layer13_bn,float *Res_Layer13_basic,float *Res_mean13,float *Res_var13,float *Res_Layer13_Gamma,float *Res_Layer13_Beta)
{
    dim3 Block4_Block(256,2,2);
    dim3 Block_Thread(7,7);
	conv<<<Block4_Block,Block_Thread>>>(NULL,Res_Layer13_Neurons,Res_Layer13_Weights,Res_Layer13_bn,14,14,1,1,3,256,false,false); 
	batchnorm<<<Block4_Block,Block_Thread>>>(Res_Layer13_bn,Res_Layer13_basic,Res_mean13,Res_var13,Res_Layer13_Gamma,Res_Layer13_Beta,14,false);
}

void res_thirteenth_basic(float *Res_Layer12_Neurons,float *Res_Layer13_basic,float *Res_Layer14_Neurons)
{
    dim3 Block4_Block(256,2,2);
    dim3 Block_Thread(7,7);
	basic_block<<<Block4_Block,Block_Thread>>>(Res_Layer12_Neurons,Res_Layer13_basic,Res_Layer14_Neurons,14,true);
}

void res_fourteenth_conv(float *Res_Layer14_Neurons,float *Res_Layer14_Weights,float *Res_Layer14_bn,float *Res_Layer15_Neurons,float *Res_mean14,float *Res_var14,float *Res_Layer14_Gamma,float *Res_Layer14_Beta)
{
	dim3 Block5_Block(512,1,1);    
    dim3 Block_Thread(7,7);
    conv<<<Block5_Block,Block_Thread>>>(NULL,Res_Layer14_Neurons,Res_Layer14_Weights,Res_Layer14_bn,14,7,2,1,3,256,false,false);
	batchnorm<<<Block5_Block,Block_Thread>>>(Res_Layer14_bn,Res_Layer15_Neurons,Res_mean14,Res_var14,Res_Layer14_Gamma,Res_Layer14_Beta,7,true);
}

void res_fifteenth_conv(float *Res_Layer15_Neurons,float *Res_Layer15_Weights,float *Res_Layer15_bn,float *Res_Layer15_basic,float *Res_mean15,float *Res_var15,float *Res_Layer15_Gamma,float *Res_Layer15_Beta)
{
    dim3 Block5_Block(512,1,1);    
    dim3 Block_Thread(7,7);
	conv<<<Block5_Block,Block_Thread>>>(NULL,Res_Layer15_Neurons,Res_Layer15_Weights,Res_Layer15_bn,7,7,1,1,3,512,false,false);
	batchnorm<<<Block5_Block,Block_Thread>>>(Res_Layer15_bn,Res_Layer15_basic,Res_mean15,Res_var15,Res_Layer15_Gamma,Res_Layer15_Beta,7,false);
}

void res_Block_D_conv(float *Res_Layer14_Neurons,float *Res_Block5_Weights,float *Res_Block5_bn,float *Res_Block5_basic,float *Res_Block5_mean,float *Res_Block5_var,float *Res_Block5_Gamma,float *Res_Block5_Beta)
{
	dim3 Block5_Block(512,1,1);    
    dim3 Block_Thread(7,7);
	conv<<<Block5_Block,Block_Thread>>>(NULL,Res_Layer14_Neurons,Res_Block5_Weights,Res_Block5_bn,14,7,2,0,1,256,false,false);
	batchnorm<<<Block5_Block,Block_Thread>>>(Res_Block5_bn,Res_Block5_basic,Res_Block5_mean,Res_Block5_var,Res_Block5_Gamma,Res_Block5_Beta,7,false);
}

void res_Block_D_basic(float *Res_Layer15_basic,float *Res_Block5_basic,float *Res_Layer16_Neurons)
{
	dim3 Block5_Block(512,1,1);    
    dim3 Block_Thread(7,7);
	basic_block<<<Block5_Block,Block_Thread>>>(Res_Layer15_basic,Res_Block5_basic,Res_Layer16_Neurons,7,true);
}

void res_sixteenth_conv(float *Res_Layer16_Neurons,float *Res_Layer16_Weights,float *Res_Layer16_bn,float *Res_Layer17_Neurons,float *Res_mean16,float *Res_var16,float *Res_Layer16_Gamma,float *Res_Layer16_Beta)
{
	dim3 Block5_Block(512,1,1);    
    dim3 Block_Thread(7,7);
	conv<<<Block5_Block,Block_Thread>>>(NULL,Res_Layer16_Neurons,Res_Layer16_Weights,Res_Layer16_bn,7,7,1,1,3,512,false,false);
	batchnorm<<<Block5_Block,Block_Thread>>>(Res_Layer16_bn,Res_Layer17_Neurons,Res_mean16,Res_var16,Res_Layer16_Gamma,Res_Layer16_Beta,7,true);
}

void res_seventeenth_conv(float *Res_Layer17_Neurons,float *Res_Layer17_Weights,float *Res_Layer17_bn,float *Res_Layer17_basic,float *Res_mean17,float *Res_var17,float *Res_Layer17_Gamma,float *Res_Layer17_Beta)
{
	dim3 Block5_Block(512,1,1);    
    dim3 Block_Thread(7,7);
	conv<<<Block5_Block,Block_Thread>>>(NULL,Res_Layer17_Neurons,Res_Layer17_Weights,Res_Layer17_bn,7,7,1,1,3,512,false,false); 
	batchnorm<<<Block5_Block,Block_Thread>>>(Res_Layer17_bn,Res_Layer17_basic,Res_mean17,Res_var17,Res_Layer17_Gamma,Res_Layer17_Beta,7,false);
}

void res_seventeenth_basic(float *Res_Layer16_Neurons,float *Res_Layer17_basic,float *Res_Layer18_Neurons)
{
	dim3 Block5_Block(512,1,1);    
    dim3 Block_Thread(7,7);
	basic_block<<<Block5_Block,Block_Thread>>>(Res_Layer16_Neurons,Res_Layer17_basic,Res_Layer18_Neurons,7,true);
}

void res_avg_pool(float *Res_Layer18_Neurons,float *Res_FC_Neurons)
{
    dim3 Avg_Block(512,1,1);
    dim3 Single_Thread(1,1);
	globalavg<<<Avg_Block,Single_Thread>>>(Res_Layer18_Neurons,Res_FC_Neurons,7);
}

void res_fc(float *Res_FC_bias,float *Res_FC_Neurons,float *Res_FC_Weights,float *Res_Result_Neurons)
{
	dim3 FC_Block(1000,1,1);
    dim3 Single_Thread(1,1);
	fc<<<FC_Block,Single_Thread>>>(Res_FC_bias,Res_FC_Neurons,Res_FC_Weights,Res_Result_Neurons,512,false);

    float *Res_Result_Neurons_CPU = (float *) malloc ((1000) * sizeof(float));
	hipMemcpy(Res_Result_Neurons_CPU, Res_Result_Neurons, (1000) * sizeof(float), hipMemcpyDeviceToHost);

	float max1 = 0.0;
	int index1 = 0; 
	for(int i = 0; i < 1000; i++){
		if(max1 < Res_Result_Neurons_CPU[i]){
			max1 = Res_Result_Neurons_CPU[i];	
			index1 = i;
		}
	}
	
	int line_count1 = 0;
	char buffer[1000];
	FILE *list1 = fopen("imagenet1000_clsidx_to_labels.txt","rt");
	while(fgets(buffer, 1000, list1) != NULL){
		line_count1++;
		if(line_count1 == (index1+1)){
			// printf("\n---Resnet18 Result---");
			// printf("\nClass ID: %d\nClass Name: %sProbability: %f\n", index1, buffer, max1);
			printf("\nResnet18: %d, %s", index1, buffer);
			break;
		}
	}
	fclose(list1);

	free(Res_Result_Neurons_CPU);	
}

void free_resnet18(float *Res_Layer1_Neurons,float *Res_Layer2_Neurons,float *Res_Layer3_Neurons,float *Res_Layer4_Neurons,
					float *Res_Layer5_Neurons,float *Res_Layer6_Neurons,float *Res_Layer7_Neurons,float *Res_Layer8_Neurons,
					float *Res_Layer9_Neurons,float *Res_Layer10_Neurons,float *Res_Layer11_Neurons,float *Res_Layer12_Neurons,
					float *Res_Layer13_Neurons,float *Res_Layer14_Neurons,float *Res_Layer15_Neurons,float *Res_Layer16_Neurons,
					float *Res_Layer17_Neurons,float *Res_Layer18_Neurons,
                    float *Res_Layer1_Weights,float *Res_Layer2_Weights,float *Res_Layer3_Weights,float *Res_Layer4_Weights,
                    float *Res_Layer5_Weights,float *Res_Layer6_Weights,float *Res_Layer7_Weights,float *Res_Layer8_Weights,
                    float *Res_Layer9_Weights,float *Res_Layer10_Weights,float *Res_Layer11_Weights,float *Res_Layer12_Weights,
                    float *Res_Layer13_Weights,float *Res_Layer14_Weights,float *Res_Layer15_Weights,float *Res_Layer16_Weights,
                    float *Res_Layer17_Weights,float *Res_Block3_Weights,float *Res_Block4_Weights,float *Res_Block5_Weights,
                    float *Res_Layer1_Gamma,float *Res_Layer2_Gamma,float *Res_Layer3_Gamma,float *Res_Layer4_Gamma,
                    float *Res_Layer5_Gamma,float *Res_Layer6_Gamma,float *Res_Layer7_Gamma,float *Res_Layer8_Gamma,
                    float *Res_Layer9_Gamma,float *Res_Layer10_Gamma,float *Res_Layer11_Gamma,float *Res_Layer12_Gamma,
                    float *Res_Layer13_Gamma,float *Res_Layer14_Gamma,float *Res_Layer15_Gamma,float *Res_Layer16_Gamma,
                    float *Res_Layer17_Gamma,float *Res_Block3_Gamma,float *Res_Block4_Gamma,float *Res_Block5_Gamma,
                    float *Res_Layer1_Beta,float *Res_Layer2_Beta,float *Res_Layer3_Beta,float *Res_Layer4_Beta,
                    float *Res_Layer5_Beta,float *Res_Layer6_Beta,float *Res_Layer7_Beta,float *Res_Layer8_Beta,
                    float *Res_Layer9_Beta,float *Res_Layer10_Beta,float *Res_Layer11_Beta,float *Res_Layer12_Beta,
                    float *Res_Layer13_Beta,float *Res_Layer14_Beta,float *Res_Layer15_Beta,float *Res_Layer16_Beta,
                    float *Res_Layer17_Beta,float *Res_Block3_Beta,float *Res_Block4_Beta,float *Res_Block5_Beta,
                    float *Res_mean1,float *Res_mean2,float *Res_mean3,float *Res_mean4,float *Res_mean5,
                    float *Res_mean6,float *Res_mean7,float *Res_mean8,float *Res_mean9,float *Res_mean10,
                    float *Res_mean11,float *Res_mean12,float *Res_mean13,float *Res_mean14,float *Res_mean15,
                    float *Res_mean16,float *Res_mean17,float *Res_Block3_mean,float *Res_Block4_mean,float *Res_Block5_mean,
                    float *Res_var1,float *Res_var2,float *Res_var3,float *Res_var4,float *Res_var5,
                    float *Res_var6,float *Res_var7,float *Res_var8,float *Res_var9,float *Res_var10,
                    float *Res_var11,float *Res_var12,float *Res_var13,float *Res_var14,float *Res_var15,
                    float *Res_var16,float *Res_var17,float *Res_Block3_var,float *Res_Block4_var,float *Res_Block5_var,
                    float *Res_FC_bias,float *Res_FC_Weights,
					float *Res_Layer3_basic,float *Res_Layer5_basic,float *Res_Layer7_basic,float *Res_Layer9_basic,
					float *Res_Layer11_basic,float *Res_Layer13_basic,float *Res_Layer15_basic,float *Res_Layer17_basic,
					float *Res_Block3_basic,float *Res_Block4_basic,float *Res_Block5_basic,
					float *Res_Layer1_bn,float *Res_Layer2_bn,float *Res_Layer3_bn,float *Res_Layer4_bn,
					float *Res_Layer5_bn,float *Res_Layer6_bn,float *Res_Layer7_bn,float *Res_Layer8_bn,
					float *Res_Layer9_bn,float *Res_Layer10_bn,float *Res_Layer11_bn,float *Res_Layer12_bn,
					float *Res_Layer13_bn,float *Res_Layer14_bn,float *Res_Layer15_bn,float *Res_Layer16_bn,
					float *Res_Layer17_bn,float *Res_Block3_bn,float *Res_Block4_bn,float *Res_Block5_bn,
					float *Res_Layer1_pool,float *Res_FC_Neurons,float *Res_Result_Neurons)
{
	hipFree(Res_Layer1_Neurons);
    hipFree(Res_Layer2_Neurons);
	hipFree(Res_Layer3_Neurons);
	hipFree(Res_Layer4_Neurons);
	hipFree(Res_Layer5_Neurons);
	hipFree(Res_Layer6_Neurons);
	hipFree(Res_Layer7_Neurons);
	hipFree(Res_Layer8_Neurons);
	hipFree(Res_Layer9_Neurons);
	hipFree(Res_Layer10_Neurons);
	hipFree(Res_Layer11_Neurons);
	hipFree(Res_Layer12_Neurons);
	hipFree(Res_Layer13_Neurons);
	hipFree(Res_Layer14_Neurons);
	hipFree(Res_Layer15_Neurons);
	hipFree(Res_Layer16_Neurons);
	hipFree(Res_Layer17_Neurons);
	hipFree(Res_Layer18_Neurons);

	hipFree(Res_Layer1_Weights);
	hipFree(Res_Layer2_Weights);
	hipFree(Res_Layer3_Weights);
	hipFree(Res_Layer4_Weights);
	hipFree(Res_Layer5_Weights);
	hipFree(Res_Layer6_Weights);
	hipFree(Res_Layer7_Weights);
	hipFree(Res_Layer8_Weights);
	hipFree(Res_Layer9_Weights);
	hipFree(Res_Layer10_Weights);
	hipFree(Res_Layer11_Weights);
	hipFree(Res_Layer12_Weights);
	hipFree(Res_Layer13_Weights);
	hipFree(Res_Layer14_Weights);
	hipFree(Res_Layer15_Weights);
	hipFree(Res_Layer16_Weights);
	hipFree(Res_Layer17_Weights);
	hipFree(Res_Block3_Weights);
	hipFree(Res_Block4_Weights);
	hipFree(Res_Block5_Weights);

	hipFree(Res_Layer1_Gamma);
	hipFree(Res_Layer2_Gamma);
	hipFree(Res_Layer3_Gamma);
	hipFree(Res_Layer4_Gamma);
	hipFree(Res_Layer5_Gamma);
	hipFree(Res_Layer6_Gamma);
	hipFree(Res_Layer7_Gamma);
	hipFree(Res_Layer8_Gamma);
	hipFree(Res_Layer9_Gamma);
	hipFree(Res_Layer10_Gamma);
	hipFree(Res_Layer11_Gamma);
	hipFree(Res_Layer12_Gamma);
	hipFree(Res_Layer13_Gamma);
	hipFree(Res_Layer14_Gamma);
	hipFree(Res_Layer15_Gamma);
	hipFree(Res_Layer16_Gamma);
	hipFree(Res_Layer17_Gamma);
	hipFree(Res_Block3_Gamma);
	hipFree(Res_Block4_Gamma);
	hipFree(Res_Block5_Gamma);

	hipFree(Res_Layer1_Beta);
	hipFree(Res_Layer2_Beta);
	hipFree(Res_Layer3_Beta);
	hipFree(Res_Layer4_Beta);
	hipFree(Res_Layer5_Beta);
	hipFree(Res_Layer6_Beta);
	hipFree(Res_Layer7_Beta);
	hipFree(Res_Layer8_Beta);
	hipFree(Res_Layer9_Beta);
	hipFree(Res_Layer10_Beta);
	hipFree(Res_Layer11_Beta);
	hipFree(Res_Layer12_Beta);
	hipFree(Res_Layer13_Beta);
	hipFree(Res_Layer14_Beta);
	hipFree(Res_Layer15_Beta);
	hipFree(Res_Layer16_Beta);
	hipFree(Res_Layer17_Beta);
	hipFree(Res_Block3_Beta);
	hipFree(Res_Block4_Beta);
	hipFree(Res_Block5_Beta);

	hipFree(Res_mean1);
	hipFree(Res_mean2);
	hipFree(Res_mean3);
	hipFree(Res_mean4);
	hipFree(Res_mean5);
	hipFree(Res_mean6);
	hipFree(Res_mean7);
	hipFree(Res_mean8);
	hipFree(Res_mean9);
	hipFree(Res_mean10);
	hipFree(Res_mean11);
	hipFree(Res_mean12);
	hipFree(Res_mean13);
	hipFree(Res_mean14);
	hipFree(Res_mean15);
	hipFree(Res_mean16);
	hipFree(Res_mean17);
	hipFree(Res_Block3_mean);
	hipFree(Res_Block4_mean);
	hipFree(Res_Block5_mean);

	hipFree(Res_var1);
	hipFree(Res_var2);
	hipFree(Res_var3);
	hipFree(Res_var4);
	hipFree(Res_var5);
	hipFree(Res_var6);
	hipFree(Res_var7);
	hipFree(Res_var8);
	hipFree(Res_var9);
	hipFree(Res_var10);
	hipFree(Res_var11);
	hipFree(Res_var12);
	hipFree(Res_var13);
	hipFree(Res_var14);
	hipFree(Res_var15);
	hipFree(Res_var16);
	hipFree(Res_var17);
	hipFree(Res_Block3_var);
	hipFree(Res_Block4_var);
	hipFree(Res_Block5_var);

	hipFree(Res_FC_bias);
	hipFree(Res_FC_Weights);

	hipFree(Res_Layer3_basic);
	hipFree(Res_Layer5_basic);
	hipFree(Res_Layer7_basic);
	hipFree(Res_Layer9_basic);
	hipFree(Res_Layer11_basic);
	hipFree(Res_Layer13_basic);
	hipFree(Res_Layer15_basic);
	hipFree(Res_Layer17_basic);
	hipFree(Res_Block3_basic);
	hipFree(Res_Block4_basic);
	hipFree(Res_Block5_basic);
	hipFree(Res_Layer1_bn);
	hipFree(Res_Layer2_bn);
	hipFree(Res_Layer3_bn);
	hipFree(Res_Layer4_bn);
	hipFree(Res_Layer5_bn);
	hipFree(Res_Layer6_bn);
	hipFree(Res_Layer7_bn);
	hipFree(Res_Layer8_bn);
	hipFree(Res_Layer9_bn);
	hipFree(Res_Layer10_bn);
	hipFree(Res_Layer11_bn);
	hipFree(Res_Layer12_bn);
	hipFree(Res_Layer13_bn);
	hipFree(Res_Layer14_bn);
	hipFree(Res_Layer15_bn);
	hipFree(Res_Layer16_bn);
	hipFree(Res_Layer17_bn);
	hipFree(Res_Block3_bn);
	hipFree(Res_Block4_bn);
	hipFree(Res_Block5_bn);
	hipFree(Res_Layer1_pool);
	hipFree(Res_FC_Neurons);
	hipFree(Res_Result_Neurons);
}

void host2gpu_vgg16(float **Vgg_Layer1_Neurons,float **Vgg_Layer2_Neurons,float **Vgg_Layer3_Neurons,float **Vgg_Layer4_Neurons,
					float **Vgg_Layer5_Neurons,float **Vgg_Layer6_Neurons,float **Vgg_Layer7_Neurons,float **Vgg_Layer8_Neurons,
					float **Vgg_Layer9_Neurons,float **Vgg_Layer10_Neurons,float **Vgg_Layer11_Neurons,float **Vgg_Layer12_Neurons,
					float **Vgg_Layer13_Neurons,float **Vgg_Layer14_Neurons,float **Vgg_Layer15_Neurons,float **Vgg_Layer16_Neurons,
                    float **Vgg_Layer1_bias,float **Vgg_Layer2_bias,float **Vgg_Layer3_bias,float **Vgg_Layer4_bias,
                    float **Vgg_Layer5_bias,float **Vgg_Layer6_bias,float **Vgg_Layer7_bias,float **Vgg_Layer8_bias,
                    float **Vgg_Layer9_bias,float **Vgg_Layer10_bias,float **Vgg_Layer11_bias,float **Vgg_Layer12_bias,
                    float **Vgg_Layer13_bias,float **Vgg_Layer14_bias,float **Vgg_Layer15_bias,float **Vgg_Layer16_bias,
                    float **Vgg_Layer1_Weights,float **Vgg_Layer2_Weights,float **Vgg_Layer3_Weights,float **Vgg_Layer4_Weights,
                    float **Vgg_Layer5_Weights,float **Vgg_Layer6_Weights,float **Vgg_Layer7_Weights,float **Vgg_Layer8_Weights,
                    float **Vgg_Layer9_Weights,float **Vgg_Layer10_Weights,float **Vgg_Layer11_Weights,float **Vgg_Layer12_Weights,
                    float **Vgg_Layer13_Weights,float **Vgg_Layer14_Weights,float **Vgg_Layer15_Weights,float **Vgg_Layer16_Weights,
                    float **Vgg_Layer2_pool,float **Vgg_Layer4_pool,float **Vgg_Layer7_pool,float **Vgg_Layer10_pool,
					float **Vgg_Layer13_pool,float **Vgg_Result_Neurons)
{
	float *Vgg_Layer1_Neurons_CPU = (float*) malloc (INPUT_SIZE * sizeof(float));
	read_parameter("data_vgg16/input_cat.txt", Vgg_Layer1_Neurons_CPU);

	float *Vgg_Layer1_bias_CPU = (float*) malloc (64 * sizeof(float)); //64
	float *Vgg_Layer2_bias_CPU = (float*) malloc (64 * sizeof(float)); //64
	float *Vgg_Layer3_bias_CPU = (float*) malloc (128 * sizeof(float)); //128
	float *Vgg_Layer4_bias_CPU = (float*) malloc (128 * sizeof(float)); //128
	float *Vgg_Layer5_bias_CPU = (float*) malloc (256 * sizeof(float)); //256
	float *Vgg_Layer6_bias_CPU = (float*) malloc (256 * sizeof(float)); //256
	float *Vgg_Layer7_bias_CPU = (float*) malloc (256 * sizeof(float)); //256
	float *Vgg_Layer8_bias_CPU = (float*) malloc (512 * sizeof(float)); //512
    float *Vgg_Layer9_bias_CPU = (float*) malloc (512 * sizeof(float)); //512
	float *Vgg_Layer10_bias_CPU = (float*) malloc (512 * sizeof(float)); //512
	float *Vgg_Layer11_bias_CPU = (float*) malloc (512 * sizeof(float)); //512
	float *Vgg_Layer12_bias_CPU = (float*) malloc (512 * sizeof(float)); //512
	float *Vgg_Layer13_bias_CPU = (float*) malloc (512 * sizeof(float)); //512
	float *Vgg_Layer14_bias_CPU = (float*) malloc (4096 * sizeof(float)); //4096
	float *Vgg_Layer15_bias_CPU = (float*) malloc (4096 * sizeof(float)); //4096
	float *Vgg_Layer16_bias_CPU = (float*) malloc (1000 * sizeof(float)); //1000

	float *Vgg_Layer1_Weights_CPU = (float*) malloc (64*3*3*3 * sizeof(float)); //64*3*3*3 = 1,728
	float *Vgg_Layer2_Weights_CPU = (float*) malloc (64*3*3*64 * sizeof(float)); //64*3*3*64 = 36,864
	float *Vgg_Layer3_Weights_CPU = (float*) malloc (128*3*3*64 * sizeof(float)); //128*3*3*64 = 73,728
	float *Vgg_Layer4_Weights_CPU = (float*) malloc (128*3*3*128 * sizeof(float)); //128*3*3*128 = 147,456
	float *Vgg_Layer5_Weights_CPU = (float*) malloc (256*3*3*128 * sizeof(float)); //256*3*3*128 = 294,912
	float *Vgg_Layer6_Weights_CPU = (float*) malloc (256*3*3*256 * sizeof(float)); //256*3*3*256 = 589,824
	float *Vgg_Layer7_Weights_CPU = (float*) malloc (256*3*3*256 * sizeof(float)); //256*3*3*256 = 589,824
	float *Vgg_Layer8_Weights_CPU = (float*) malloc (512*3*3*256 * sizeof(float)); //512*3*3*256 = 1,179,648
    float *Vgg_Layer9_Weights_CPU = (float*) malloc (512*3*3*512 * sizeof(float)); //512*3*3*512 = 2,359,296
	float *Vgg_Layer10_Weights_CPU = (float*) malloc (512*3*3*512 * sizeof(float)); //512*3*3*512 = 2,359,296
	float *Vgg_Layer11_Weights_CPU = (float*) malloc (512*3*3*512 * sizeof(float)); //512*3*3*512 = 2,359,296
	float *Vgg_Layer12_Weights_CPU = (float*) malloc (512*3*3*512 * sizeof(float)); //512*3*3*512 = 2,359,296
	float *Vgg_Layer13_Weights_CPU = (float*) malloc (512*3*3*512 * sizeof(float)); //512*3*3*512 = 2,359,296
	float *Vgg_Layer14_Weights_CPU = (float*) malloc (4096*512*7*7 * sizeof(float)); //4096*512*7*7 = 102,760,448
	float *Vgg_Layer15_Weights_CPU = (float*) malloc (4096*4096 * sizeof(float)); //4096*4096 = 16,777,216
	float *Vgg_Layer16_Weights_CPU = (float*) malloc (1000*4096 * sizeof(float)); //1000*4096 = 4,096,000

	read_parameter("data_vgg16/bias1.txt", Vgg_Layer1_bias_CPU);
	read_parameter("data_vgg16/bias2.txt", Vgg_Layer2_bias_CPU);
	read_parameter("data_vgg16/bias3.txt", Vgg_Layer3_bias_CPU);
	read_parameter("data_vgg16/bias4.txt", Vgg_Layer4_bias_CPU);
	read_parameter("data_vgg16/bias5.txt", Vgg_Layer5_bias_CPU);
	read_parameter("data_vgg16/bias6.txt", Vgg_Layer6_bias_CPU);
	read_parameter("data_vgg16/bias7.txt", Vgg_Layer7_bias_CPU);
	read_parameter("data_vgg16/bias8.txt", Vgg_Layer8_bias_CPU);
    read_parameter("data_vgg16/bias9.txt", Vgg_Layer9_bias_CPU);
	read_parameter("data_vgg16/bias10.txt", Vgg_Layer10_bias_CPU);
	read_parameter("data_vgg16/bias11.txt", Vgg_Layer11_bias_CPU);
	read_parameter("data_vgg16/bias12.txt", Vgg_Layer12_bias_CPU);
	read_parameter("data_vgg16/bias13.txt", Vgg_Layer13_bias_CPU);
	read_parameter("data_vgg16/bias14.txt", Vgg_Layer14_bias_CPU);
	read_parameter("data_vgg16/bias15.txt", Vgg_Layer15_bias_CPU);
	read_parameter("data_vgg16/bias16.txt", Vgg_Layer16_bias_CPU);

	read_parameter("data_vgg16/conv1.txt", Vgg_Layer1_Weights_CPU);
	read_parameter("data_vgg16/conv2.txt", Vgg_Layer2_Weights_CPU);
	read_parameter("data_vgg16/conv3.txt", Vgg_Layer3_Weights_CPU);
	read_parameter("data_vgg16/conv4.txt", Vgg_Layer4_Weights_CPU);
	read_parameter("data_vgg16/conv5.txt", Vgg_Layer5_Weights_CPU);
	read_parameter("data_vgg16/conv6.txt", Vgg_Layer6_Weights_CPU);
	read_parameter("data_vgg16/conv7.txt", Vgg_Layer7_Weights_CPU);
	read_parameter("data_vgg16/conv8.txt", Vgg_Layer8_Weights_CPU);
 	read_parameter("data_vgg16/conv9.txt", Vgg_Layer9_Weights_CPU);
	read_parameter("data_vgg16/conv10.txt", Vgg_Layer10_Weights_CPU);
	read_parameter("data_vgg16/conv11.txt", Vgg_Layer11_Weights_CPU);
	read_parameter("data_vgg16/conv12.txt", Vgg_Layer12_Weights_CPU);
	read_parameter("data_vgg16/conv13.txt", Vgg_Layer13_Weights_CPU);
	read_parameter("data_vgg16/fc14.txt", Vgg_Layer14_Weights_CPU);
	read_parameter("data_vgg16/fc15.txt", Vgg_Layer15_Weights_CPU);
	read_parameter("data_vgg16/fc16.txt", Vgg_Layer16_Weights_CPU);

    float *Vgg_Layer1_Neurons_data;
	float *Vgg_Layer1_bias_data, *Vgg_Layer2_bias_data, *Vgg_Layer3_bias_data, *Vgg_Layer4_bias_data,
			*Vgg_Layer5_bias_data, *Vgg_Layer6_bias_data, *Vgg_Layer7_bias_data, *Vgg_Layer8_bias_data,
			*Vgg_Layer9_bias_data, *Vgg_Layer10_bias_data, *Vgg_Layer11_bias_data, *Vgg_Layer12_bias_data,
			*Vgg_Layer13_bias_data, *Vgg_Layer14_bias_data, *Vgg_Layer15_bias_data, *Vgg_Layer16_bias_data;
	float *Vgg_Layer1_Weights_data, *Vgg_Layer2_Weights_data, *Vgg_Layer3_Weights_data, *Vgg_Layer4_Weights_data, 
			*Vgg_Layer5_Weights_data, *Vgg_Layer6_Weights_data, *Vgg_Layer7_Weights_data, *Vgg_Layer8_Weights_data,
			*Vgg_Layer9_Weights_data, *Vgg_Layer10_Weights_data, *Vgg_Layer11_Weights_data, *Vgg_Layer12_Weights_data, 
			*Vgg_Layer13_Weights_data, *Vgg_Layer14_Weights_data, *Vgg_Layer15_Weights_data, *Vgg_Layer16_Weights_data;
	
	hipMalloc((void**) &Vgg_Layer1_Neurons_data, INPUT_SIZE * sizeof(float)); //224*224*3
	hipMalloc((void**) &Vgg_Layer1_bias_data, 64 * sizeof(float)); //64
	hipMalloc((void**) &Vgg_Layer1_Weights_data, (64*(3*3*3)) * sizeof(float)); //64*3*3*3 = 1728
	hipMalloc((void**) &Vgg_Layer2_bias_data, 64 * sizeof(float)); //64
	hipMalloc((void**) &Vgg_Layer2_Weights_data, (64*(3*3*64)) * sizeof(float)); //64*3*3*64 = 36864
	hipMalloc((void**) &Vgg_Layer3_bias_data, 128 * sizeof(float)); //128
	hipMalloc((void**) &Vgg_Layer3_Weights_data, (128*(3*3*64)) * sizeof(float)); //128*3*3*64 = 73728
	hipMalloc((void**) &Vgg_Layer4_bias_data, 128 * sizeof(float)); //128
	hipMalloc((void**) &Vgg_Layer4_Weights_data, (128*(3*3*128)) * sizeof(float)); //128*3*3*128 = 147456
	hipMalloc((void**) &Vgg_Layer5_bias_data, 256 * sizeof(float)); //256
	hipMalloc((void**) &Vgg_Layer5_Weights_data, (256*(3*3*128)) * sizeof(float)); //256*3*3*128 = 294912
	hipMalloc((void**) &Vgg_Layer6_bias_data, 256 * sizeof(float)); //256
	hipMalloc((void**) &Vgg_Layer6_Weights_data, (256*(3*3*256)) * sizeof(float)); //256*3*3*256 = 589824
	hipMalloc((void**) &Vgg_Layer7_bias_data, 256 * sizeof(float)); //256
	hipMalloc((void**) &Vgg_Layer7_Weights_data, (256*(3*3*256)) * sizeof(float)); //256*3*3*256 = 589824
	hipMalloc((void**) &Vgg_Layer8_bias_data, 512 * sizeof(float)); //512
	hipMalloc((void**) &Vgg_Layer8_Weights_data, (512*(3*3*256)) * sizeof(float)); //512*3*3*256 = 1179648
	hipMalloc((void**) &Vgg_Layer9_bias_data, 512 * sizeof(float)); //512
	hipMalloc((void**) &Vgg_Layer9_Weights_data, (512*(3*3*512)) * sizeof(float)); //512*3*3*512 = 2359296
	hipMalloc((void**) &Vgg_Layer10_bias_data, 512 * sizeof(float)); //512
	hipMalloc((void**) &Vgg_Layer10_Weights_data, (512*(3*3*512)) * sizeof(float)); //512*3*3*512 = 2359296
	hipMalloc((void**) &Vgg_Layer11_bias_data, 512 * sizeof(float)); //512
	hipMalloc((void**) &Vgg_Layer11_Weights_data, (512*(3*3*512)) * sizeof(float)); //512*3*3*512 = 2359296
	hipMalloc((void**) &Vgg_Layer12_bias_data, 512 * sizeof(float)); //512
	hipMalloc((void**) &Vgg_Layer12_Weights_data, (512*(3*3*512)) * sizeof(float)); //512*3*3*512 = 2359296
	hipMalloc((void**) &Vgg_Layer13_bias_data, 512 * sizeof(float)); //256
	hipMalloc((void**) &Vgg_Layer13_Weights_data, (512*(3*3*512)) * sizeof(float)); //512*3*3*512 = 2359296
	hipMalloc((void**) &Vgg_Layer14_bias_data, 4096 * sizeof(float)); //4096
	hipMalloc((void**) &Vgg_Layer14_Weights_data, (4096*(512*(7*7))) * sizeof(float)); //4096*512*7*7
	hipMalloc((void**) &Vgg_Layer15_bias_data, 4096 * sizeof(float)); //4096
	hipMalloc((void**) &Vgg_Layer15_Weights_data, (4096*4096) * sizeof(float)); //4096*4096
	hipMalloc((void**) &Vgg_Layer16_bias_data, 1000 * sizeof(float)); //1000
	hipMalloc((void**) &Vgg_Layer16_Weights_data, (1000*4096) * sizeof(float)); //1000*4096

	hipMemcpy(Vgg_Layer1_Neurons_data, Vgg_Layer1_Neurons_CPU, INPUT_SIZE * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(Vgg_Layer1_bias_data, Vgg_Layer1_bias_CPU, 64 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(Vgg_Layer1_Weights_data, Vgg_Layer1_Weights_CPU, (64*(3*3*3)) * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(Vgg_Layer2_bias_data, Vgg_Layer2_bias_CPU, 64 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(Vgg_Layer2_Weights_data, Vgg_Layer2_Weights_CPU, (64*(3*3*64)) * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(Vgg_Layer3_bias_data, Vgg_Layer3_bias_CPU, 128 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(Vgg_Layer3_Weights_data, Vgg_Layer3_Weights_CPU, (128*(3*3*64)) * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(Vgg_Layer4_bias_data, Vgg_Layer4_bias_CPU, 64 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(Vgg_Layer4_Weights_data, Vgg_Layer4_Weights_CPU, (64*(3*3*128)) * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(Vgg_Layer5_bias_data, Vgg_Layer5_bias_CPU, 256 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(Vgg_Layer5_Weights_data, Vgg_Layer5_Weights_CPU, (256*(3*3*128)) * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(Vgg_Layer6_bias_data, Vgg_Layer6_bias_CPU, 256 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(Vgg_Layer6_Weights_data, Vgg_Layer6_Weights_CPU, (256*(3*3*256)) * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(Vgg_Layer7_bias_data, Vgg_Layer7_bias_CPU, 256 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(Vgg_Layer7_Weights_data, Vgg_Layer7_Weights_CPU, (256*(3*3*256)) * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(Vgg_Layer8_bias_data, Vgg_Layer8_bias_CPU, 512 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(Vgg_Layer8_Weights_data, Vgg_Layer8_Weights_CPU, (512*(3*3*256)) * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(Vgg_Layer9_bias_data, Vgg_Layer9_bias_CPU, 512 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(Vgg_Layer9_Weights_data, Vgg_Layer9_Weights_CPU, (512*(3*3*512)) * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(Vgg_Layer10_bias_data, Vgg_Layer10_bias_CPU, 512 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(Vgg_Layer10_Weights_data, Vgg_Layer10_Weights_CPU, (512*(3*3*512)) * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(Vgg_Layer11_bias_data, Vgg_Layer11_bias_CPU, 512 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(Vgg_Layer11_Weights_data, Vgg_Layer11_Weights_CPU, (512*(3*3*512)) * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(Vgg_Layer12_bias_data, Vgg_Layer12_bias_CPU, 512 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(Vgg_Layer12_Weights_data, Vgg_Layer12_Weights_CPU, (512*(3*3*512)) * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(Vgg_Layer13_bias_data, Vgg_Layer13_bias_CPU, 512 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(Vgg_Layer13_Weights_data, Vgg_Layer13_Weights_CPU, (512*(3*3*512)) * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(Vgg_Layer14_bias_data, Vgg_Layer14_bias_CPU, 4096 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(Vgg_Layer14_Weights_data, Vgg_Layer14_Weights_CPU, (4096*(512*(7*7))) * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(Vgg_Layer15_bias_data, Vgg_Layer15_bias_CPU, 4096 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(Vgg_Layer15_Weights_data, Vgg_Layer15_Weights_CPU, (4096*4096) * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(Vgg_Layer16_bias_data, Vgg_Layer16_bias_CPU, 1000 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(Vgg_Layer16_Weights_data, Vgg_Layer16_Weights_CPU, (1000*4096) * sizeof(float), hipMemcpyHostToDevice);

	*Vgg_Layer1_Neurons = Vgg_Layer1_Neurons_data;

	*Vgg_Layer1_bias = Vgg_Layer1_bias_data;
	*Vgg_Layer2_bias = Vgg_Layer2_bias_data;
	*Vgg_Layer3_bias = Vgg_Layer3_bias_data;
	*Vgg_Layer4_bias = Vgg_Layer4_bias_data;
	*Vgg_Layer5_bias = Vgg_Layer5_bias_data;
	*Vgg_Layer6_bias = Vgg_Layer6_bias_data;
	*Vgg_Layer7_bias = Vgg_Layer7_bias_data;
	*Vgg_Layer8_bias = Vgg_Layer8_bias_data;
	*Vgg_Layer9_bias = Vgg_Layer9_bias_data;
	*Vgg_Layer10_bias = Vgg_Layer10_bias_data;
	*Vgg_Layer11_bias = Vgg_Layer11_bias_data;
	*Vgg_Layer12_bias = Vgg_Layer12_bias_data;
	*Vgg_Layer13_bias = Vgg_Layer13_bias_data;
	*Vgg_Layer14_bias = Vgg_Layer14_bias_data;
	*Vgg_Layer15_bias = Vgg_Layer15_bias_data;
	*Vgg_Layer16_bias = Vgg_Layer16_bias_data;

	*Vgg_Layer1_Weights = Vgg_Layer1_Weights_data;
	*Vgg_Layer2_Weights = Vgg_Layer2_Weights_data;
	*Vgg_Layer3_Weights = Vgg_Layer3_Weights_data;
	*Vgg_Layer4_Weights = Vgg_Layer4_Weights_data;
	*Vgg_Layer5_Weights = Vgg_Layer5_Weights_data;
	*Vgg_Layer6_Weights = Vgg_Layer6_Weights_data;
	*Vgg_Layer7_Weights = Vgg_Layer7_Weights_data;
	*Vgg_Layer8_Weights = Vgg_Layer8_Weights_data;
	*Vgg_Layer9_Weights = Vgg_Layer9_Weights_data;
	*Vgg_Layer10_Weights = Vgg_Layer10_Weights_data;
	*Vgg_Layer11_Weights = Vgg_Layer11_Weights_data;
	*Vgg_Layer12_Weights = Vgg_Layer12_Weights_data;
	*Vgg_Layer13_Weights = Vgg_Layer13_Weights_data;
	*Vgg_Layer14_Weights = Vgg_Layer14_Weights_data;
	*Vgg_Layer15_Weights = Vgg_Layer15_Weights_data;
	*Vgg_Layer16_Weights = Vgg_Layer16_Weights_data;

	free(Vgg_Layer1_Neurons_CPU);
	free(Vgg_Layer1_bias_CPU);
	free(Vgg_Layer2_bias_CPU);
	free(Vgg_Layer3_bias_CPU);
	free(Vgg_Layer4_bias_CPU);
	free(Vgg_Layer5_bias_CPU);
	free(Vgg_Layer6_bias_CPU);
	free(Vgg_Layer7_bias_CPU);
	free(Vgg_Layer8_bias_CPU);
	free(Vgg_Layer9_bias_CPU);
	free(Vgg_Layer10_bias_CPU);
	free(Vgg_Layer11_bias_CPU);
	free(Vgg_Layer12_bias_CPU);
	free(Vgg_Layer13_bias_CPU);
	free(Vgg_Layer14_bias_CPU);
	free(Vgg_Layer15_bias_CPU);
	free(Vgg_Layer16_bias_CPU);

    free(Vgg_Layer1_Weights_CPU);
    free(Vgg_Layer2_Weights_CPU);
    free(Vgg_Layer3_Weights_CPU);
    free(Vgg_Layer4_Weights_CPU);
    free(Vgg_Layer5_Weights_CPU);
    free(Vgg_Layer6_Weights_CPU);
    free(Vgg_Layer7_Weights_CPU);
    free(Vgg_Layer8_Weights_CPU);
	free(Vgg_Layer9_Weights_CPU);
    free(Vgg_Layer10_Weights_CPU);
    free(Vgg_Layer11_Weights_CPU);
    free(Vgg_Layer12_Weights_CPU);
    free(Vgg_Layer13_Weights_CPU);
    free(Vgg_Layer14_Weights_CPU);
    free(Vgg_Layer15_Weights_CPU);
    free(Vgg_Layer16_Weights_CPU);

	float *Vgg_Layer2_Neurons_data; 
	hipMalloc((void**) &Vgg_Layer2_Neurons_data, (64*224*224) * sizeof(float)); //64*224*224
	*Vgg_Layer2_Neurons = Vgg_Layer2_Neurons_data;

    float *Vgg_Layer2_pool_data;
    hipMalloc((void**) &Vgg_Layer2_pool_data, (64*224*224) * sizeof(float)); //64*224*224
	*Vgg_Layer2_pool = Vgg_Layer2_pool_data;

    float *Vgg_Layer3_Neurons_data;
    hipMalloc((void**) &Vgg_Layer3_Neurons_data, (64*112*112) * sizeof(float)); //64*112*112
	*Vgg_Layer3_Neurons = Vgg_Layer3_Neurons_data;

    float *Vgg_Layer4_Neurons_data;
    hipMalloc((void**) &Vgg_Layer4_Neurons_data, (128*112*112) * sizeof(float)); //128*112*112
	*Vgg_Layer4_Neurons = Vgg_Layer4_Neurons_data;

    float *Vgg_Layer4_pool_data;
    hipMalloc((void**) &Vgg_Layer4_pool_data, (128*112*112) * sizeof(float)); //128*112*112
	*Vgg_Layer4_pool = Vgg_Layer4_pool_data;
	
    float *Vgg_Layer5_Neurons_data;
	hipMalloc((void**) &Vgg_Layer5_Neurons_data, (128*56*56) * sizeof(float)); //128*56*56
	*Vgg_Layer5_Neurons = Vgg_Layer5_Neurons_data;

    float *Vgg_Layer6_Neurons_data;
   	hipMalloc((void**) &Vgg_Layer6_Neurons_data, (256*56*56) * sizeof(float)); //256*56*56
	*Vgg_Layer6_Neurons = Vgg_Layer6_Neurons_data;

    float *Vgg_Layer7_Neurons_data;
    hipMalloc((void**) &Vgg_Layer7_Neurons_data, (256*56*56) * sizeof(float)); //256*56*56
	*Vgg_Layer7_Neurons = Vgg_Layer7_Neurons_data;

    float *Vgg_Layer7_pool_data;
    hipMalloc((void**) &Vgg_Layer7_pool_data, (256*56*56) * sizeof(float)); //256*56*56
	*Vgg_Layer7_pool = Vgg_Layer7_pool_data;

    float *Vgg_Layer8_Neurons_data;
    hipMalloc((void**) &Vgg_Layer8_Neurons_data, (256*28*28) * sizeof(float)); //256*28*28
	*Vgg_Layer8_Neurons = Vgg_Layer8_Neurons_data;

    float *Vgg_Layer9_Neurons_data;
    hipMalloc((void**) &Vgg_Layer9_Neurons_data, (512*28*28) * sizeof(float)); //512*28*28
	*Vgg_Layer9_Neurons = Vgg_Layer9_Neurons_data;

    float *Vgg_Layer10_Neurons_data;
    hipMalloc((void**) &Vgg_Layer10_Neurons_data, (512*28*28) * sizeof(float)); //512*28*28
	*Vgg_Layer10_Neurons = Vgg_Layer10_Neurons_data;

    float *Vgg_Layer10_pool_data;
    hipMalloc((void**) &Vgg_Layer10_pool_data, (512*28*28) * sizeof(float)); //512*28*28
	*Vgg_Layer10_pool = Vgg_Layer10_pool_data;

    float *Vgg_Layer11_Neurons_data;
    hipMalloc((void**) &Vgg_Layer11_Neurons_data, (512*14*14) * sizeof(float)); //512*14*14
	*Vgg_Layer11_Neurons = Vgg_Layer11_Neurons_data;

    float *Vgg_Layer12_Neurons_data;
    hipMalloc((void**) &Vgg_Layer12_Neurons_data, (512*14*14) * sizeof(float)); //512*14*14 
	*Vgg_Layer12_Neurons = Vgg_Layer12_Neurons_data;

    float *Vgg_Layer13_Neurons_data;
    hipMalloc((void**) &Vgg_Layer13_Neurons_data, (512*14*14) * sizeof(float)); //512*14*14
	*Vgg_Layer13_Neurons = Vgg_Layer13_Neurons_data;

    float *Vgg_Layer13_pool_data;
    hipMalloc((void**) &Vgg_Layer13_pool_data, (512*14*14) * sizeof(float)); //512*14*14
	*Vgg_Layer13_pool = Vgg_Layer13_pool_data;

    float *Vgg_Layer14_Neurons_data;
    hipMalloc((void**) &Vgg_Layer14_Neurons_data, (512*7*7) * sizeof(float)); //512*7*7
	*Vgg_Layer14_Neurons = Vgg_Layer14_Neurons_data;

    float *Vgg_Layer15_Neurons_data;
	hipMalloc((void**) &Vgg_Layer15_Neurons_data, 4096 * sizeof(float)); //4096
	*Vgg_Layer15_Neurons = Vgg_Layer15_Neurons_data;

    float *Vgg_Layer16_Neurons_data;
	hipMalloc((void**) &Vgg_Layer16_Neurons_data, 4096 * sizeof(float)); //4096
	*Vgg_Layer16_Neurons = Vgg_Layer16_Neurons_data;

    float *Vgg_Result_Neurons_data;
	hipMalloc((void**) &Vgg_Result_Neurons_data, 1000 * sizeof(float)); //1000
	*Vgg_Result_Neurons = Vgg_Result_Neurons_data;
}



void vgg_first_conv(float *Vgg_Layer1_bias,float *Vgg_Layer1_Neurons,float *Vgg_Layer1_Weights,float *Vgg_Layer2_Neurons)
{
	dim3 Block1_Block(64,32,32);
    dim3 Block_Thread(7,7);
	first<<<Block1_Block,Block_Thread>>>(Vgg_Layer1_bias,Vgg_Layer1_Neurons,Vgg_Layer1_Weights,Vgg_Layer2_Neurons,224,224,1,1,3,3,true,true);
}

void vgg_second_conv(float *Vgg_Layer2_bias,float *Vgg_Layer2_Neurons,float *Vgg_Layer2_Weights,float *Vgg_Layer2_pool)
{
	dim3 Block1_Block(64,32,32);
    dim3 Block_Thread(7,7);
	conv<<<Block1_Block,Block_Thread>>>(Vgg_Layer2_bias,Vgg_Layer2_Neurons,Vgg_Layer2_Weights,Vgg_Layer2_pool,224,224,1,1,3,64,true,true);
}

void vgg_second_pool(float *Vgg_Layer2_pool,float *Vgg_Layer3_Neurons)
{
	dim3 Block1_Pool_Block(64,16,16);
	dim3 Block_Thread(7,7);
    max<<<Block1_Pool_Block,Block_Thread>>>(Vgg_Layer2_pool,Vgg_Layer3_Neurons,224,112,2,0,2);
}

void vgg_third_conv(float *Vgg_Layer3_bias,float *Vgg_Layer3_Neurons,float *Vgg_Layer3_Weights,float *Vgg_Layer4_Neurons)
{
	dim3 Block2_Block(128,16,16);
	dim3 Block_Thread(7,7);
	conv<<<Block2_Block,Block_Thread>>>(Vgg_Layer3_bias,Vgg_Layer3_Neurons,Vgg_Layer3_Weights,Vgg_Layer4_Neurons,112,112,1,1,3,64,true,true);
}

void vgg_fourth_conv(float *Vgg_Layer4_bias,float *Vgg_Layer4_Neurons,float *Vgg_Layer4_Weights,float *Vgg_Layer4_pool)
{
	dim3 Block2_Block(128,16,16);
	dim3 Block_Thread(7,7);
	conv<<<Block2_Block,Block_Thread>>>(Vgg_Layer4_bias,Vgg_Layer4_Neurons,Vgg_Layer4_Weights,Vgg_Layer4_pool,112,112,1,1,3,128,true,true);
}

void vgg_fourth_pool(float *Vgg_Layer4_pool,float *Vgg_Layer5_Neurons)
{
	dim3 Block2_Pool_Block(128,8,8);
	dim3 Block_Thread(7,7);
    max<<<Block2_Pool_Block,Block_Thread>>>(Vgg_Layer4_pool,Vgg_Layer5_Neurons,112,56,2,0,2);
}

void vgg_fifth_conv(float *Vgg_Layer5_bias,float *Vgg_Layer5_Neurons,float *Vgg_Layer5_Weights,float *Vgg_Layer6_Neurons)
{
	dim3 Block3_Block(256,8,8);
	dim3 Block_Thread(7,7);
	conv<<<Block3_Block,Block_Thread>>>(Vgg_Layer5_bias,Vgg_Layer5_Neurons,Vgg_Layer5_Weights,Vgg_Layer6_Neurons,56,56,1,1,3,128,true,true);
}

void vgg_sixth_conv(float *Vgg_Layer6_bias,float *Vgg_Layer6_Neurons,float *Vgg_Layer6_Weights,float *Vgg_Layer7_Neurons)
{
	dim3 Block3_Block(256,8,8);
	dim3 Block_Thread(7,7);
	conv<<<Block3_Block,Block_Thread>>>(Vgg_Layer6_bias,Vgg_Layer6_Neurons,Vgg_Layer6_Weights,Vgg_Layer7_Neurons,56,56,1,1,3,256,true,true);
}

void vgg_seventh_conv(float *Vgg_Layer7_bias,float *Vgg_Layer7_Neurons,float *Vgg_Layer7_Weights,float *Vgg_Layer7_pool)
{
	dim3 Block3_Block(256,8,8);
	dim3 Block_Thread(7,7);
	conv<<<Block3_Block,Block_Thread>>>(Vgg_Layer7_bias,Vgg_Layer7_Neurons,Vgg_Layer7_Weights,Vgg_Layer7_pool,56,56,1,1,3,256,true,true);
}

void vgg_seventh_pool(float *Vgg_Layer7_pool,float *Vgg_Layer8_Neurons)
{
	dim3 Block3_Pool_Block(256,4,4);
	dim3 Block_Thread(7,7);
    max<<<Block3_Pool_Block,Block_Thread>>>(Vgg_Layer7_pool,Vgg_Layer8_Neurons,56,28,2,0,2);
}

void vgg_eighth_conv(float *Vgg_Layer8_bias,float *Vgg_Layer8_Neurons,float *Vgg_Layer8_Weights,float *Vgg_Layer9_Neurons)
{
	dim3 Block4_Block(512,4,4);
	dim3 Block_Thread(7,7);
	conv<<<Block4_Block,Block_Thread>>>(Vgg_Layer8_bias,Vgg_Layer8_Neurons,Vgg_Layer8_Weights,Vgg_Layer9_Neurons,28,28,1,1,3,256,true,true);
}

void vgg_ninth_conv(float *Vgg_Layer9_bias,float *Vgg_Layer9_Neurons,float *Vgg_Layer9_Weights,float *Vgg_Layer10_Neurons)
{
	dim3 Block4_Block(512,4,4);
	dim3 Block_Thread(7,7);
    conv<<<Block4_Block,Block_Thread>>>(Vgg_Layer9_bias,Vgg_Layer9_Neurons,Vgg_Layer9_Weights,Vgg_Layer10_Neurons,28,28,1,1,3,512,true,true);
}

void vgg_tenth_conv(float *Vgg_Layer10_bias,float *Vgg_Layer10_Neurons,float *Vgg_Layer10_Weights,float *Vgg_Layer10_pool)
{
	dim3 Block4_Block(512,4,4);
	dim3 Block_Thread(7,7);
    conv<<<Block4_Block,Block_Thread>>>(Vgg_Layer10_bias,Vgg_Layer10_Neurons,Vgg_Layer10_Weights,Vgg_Layer10_pool,28,28,1,1,3,512,true,true);
}

void vgg_tenth_pool(float *Vgg_Layer10_pool,float *Vgg_Layer11_Neurons)
{
	dim3 Block4_Pool_Block(512,2,2);
	dim3 Block_Thread(7,7);	
    max<<<Block4_Pool_Block,Block_Thread>>>(Vgg_Layer10_pool,Vgg_Layer11_Neurons,28,14,2,0,2);
}

void vgg_eleventh_conv(float *Vgg_Layer11_bias,float *Vgg_Layer11_Neurons,float *Vgg_Layer11_Weights,float *Vgg_Layer12_Neurons)
{
	dim3 Block5_Block(512,2,2);
	dim3 Block_Thread(7,7);
	conv<<<Block5_Block,Block_Thread>>>(Vgg_Layer11_bias,Vgg_Layer11_Neurons,Vgg_Layer11_Weights,Vgg_Layer12_Neurons,14,14,1,1,3,512,true,true);
}

void vgg_twelfth_conv(float *Vgg_Layer12_bias,float *Vgg_Layer12_Neurons,float *Vgg_Layer12_Weights,float *Vgg_Layer13_Neurons)
{
	dim3 Block5_Block(512,2,2);
	dim3 Block_Thread(7,7);
    conv<<<Block5_Block,Block_Thread>>>(Vgg_Layer12_bias,Vgg_Layer12_Neurons,Vgg_Layer12_Weights,Vgg_Layer13_Neurons,14,14,1,1,3,512,true,true);
}

void vgg_thirteenth_conv(float *Vgg_Layer13_bias,float *Vgg_Layer13_Neurons,float *Vgg_Layer13_Weights,float *Vgg_Layer13_pool)
{
	dim3 Block5_Block(512,2,2);
	dim3 Block_Thread(7,7);
    conv<<<Block5_Block,Block_Thread>>>(Vgg_Layer13_bias,Vgg_Layer13_Neurons,Vgg_Layer13_Weights,Vgg_Layer13_pool,14,14,1,1,3,512,true,true);
}

void vgg_thirteenth_pool(float *Vgg_Layer13_pool,float *Vgg_Layer14_Neurons)
{
	dim3 Block5_Pool_Block(512,1,1);
	dim3 Block_Thread(7,7);
    max<<<Block5_Pool_Block,Block_Thread>>>(Vgg_Layer13_pool,Vgg_Layer14_Neurons,14,7,2,0,2);
}

void vgg_first_fc(float *Vgg_Layer14_bias,float *Vgg_Layer14_Neurons,float *Vgg_Layer14_Weights,float *Vgg_Layer15_Neurons)
{
	dim3 FC1_Block(4096,1,1);
	dim3 FC1_Thread(1,1);
	fc<<<FC1_Block,FC1_Thread>>>(Vgg_Layer14_bias,Vgg_Layer14_Neurons,Vgg_Layer14_Weights,Vgg_Layer15_Neurons,(7*7*512),true);
}

void vgg_second_fc(float *Vgg_Layer15_bias,float *Vgg_Layer15_Neurons,float *Vgg_Layer15_Weights,float *Vgg_Layer16_Neurons)
{
	dim3 FC2_Block(4096,1,1);
	dim3 FC2_Thread(1,1);
	fc<<<FC2_Block,FC2_Thread>>>(Vgg_Layer15_bias,Vgg_Layer15_Neurons,Vgg_Layer15_Weights,Vgg_Layer16_Neurons,4096,true);
}

void vgg_third_fc(float *Vgg_Layer16_bias,float *Vgg_Layer16_Neurons,float *Vgg_Layer16_Weights,float *Vgg_Result_Neurons)
{
	dim3 FC3_Block(1000,1,1);
	dim3 FC3_Thread(1,1);
	fc<<<FC3_Block,FC3_Thread>>>(Vgg_Layer16_bias,Vgg_Layer16_Neurons,Vgg_Layer16_Weights,Vgg_Result_Neurons,4096,false);

	float *Vgg_Result_Neurons_CPU = (float *) malloc ((1000) * sizeof(float));
	hipMemcpy(Vgg_Result_Neurons_CPU, Vgg_Result_Neurons, (1000) * sizeof(float), hipMemcpyDeviceToHost);


	float max1 = 0.0;
	int index1 = 0; 
	for(int i = 0; i < 1000; i++){
		if(max1 < Vgg_Result_Neurons_CPU[i]){
			max1 = Vgg_Result_Neurons_CPU[i];	
			index1 = i;
		}
	}
	
	int line_count1 = 0;
	char buffer[1000];
	FILE *list1 = fopen("imagenet1000_clsidx_to_labels.txt","rt");
	while(fgets(buffer, 1000, list1) != NULL){
		line_count1++;
		if(line_count1 == (index1+1)){
			// printf("\n---Vgg16 Result---");
			// printf("\nClass ID: %d\nClass Name: %sProbability: %f\n", index1, buffer, max1);
			printf("\nVgg16: %d, %s", index1, buffer);
			break;
		}
	}
	fclose(list1);

	free(Vgg_Result_Neurons_CPU);
}

void free_vgg16(float *Vgg_Layer1_Neurons,float *Vgg_Layer2_Neurons,float *Vgg_Layer3_Neurons,float *Vgg_Layer4_Neurons,
					float *Vgg_Layer5_Neurons,float *Vgg_Layer6_Neurons,float *Vgg_Layer7_Neurons,float *Vgg_Layer8_Neurons,
					float *Vgg_Layer9_Neurons,float *Vgg_Layer10_Neurons,float *Vgg_Layer11_Neurons,float *Vgg_Layer12_Neurons,
					float *Vgg_Layer13_Neurons,float *Vgg_Layer14_Neurons,float *Vgg_Layer15_Neurons,float *Vgg_Layer16_Neurons,
                    float *Vgg_Layer1_bias,float *Vgg_Layer2_bias,float *Vgg_Layer3_bias,float *Vgg_Layer4_bias,
                    float *Vgg_Layer5_bias,float *Vgg_Layer6_bias,float *Vgg_Layer7_bias,float *Vgg_Layer8_bias,
                    float *Vgg_Layer9_bias,float *Vgg_Layer10_bias,float *Vgg_Layer11_bias,float *Vgg_Layer12_bias,
                    float *Vgg_Layer13_bias,float *Vgg_Layer14_bias,float *Vgg_Layer15_bias,float *Vgg_Layer16_bias,
                    float *Vgg_Layer1_Weights,float *Vgg_Layer2_Weights,float *Vgg_Layer3_Weights,float *Vgg_Layer4_Weights,
                    float *Vgg_Layer5_Weights,float *Vgg_Layer6_Weights,float *Vgg_Layer7_Weights,float *Vgg_Layer8_Weights,
                    float *Vgg_Layer9_Weights,float *Vgg_Layer10_Weights,float *Vgg_Layer11_Weights,float *Vgg_Layer12_Weights,
                    float *Vgg_Layer13_Weights,float *Vgg_Layer14_Weights,float *Vgg_Layer15_Weights,float *Vgg_Layer16_Weights,
                    float *Vgg_Layer2_pool,float *Vgg_Layer4_pool,float *Vgg_Layer7_pool,float *Vgg_Layer10_pool,
					float *Vgg_Layer13_pool,float *Vgg_Result_Neurons)
{
	hipFree(Vgg_Layer1_Neurons);
    hipFree(Vgg_Layer2_Neurons);
	hipFree(Vgg_Layer3_Neurons);
	hipFree(Vgg_Layer4_Neurons);
	hipFree(Vgg_Layer5_Neurons);
	hipFree(Vgg_Layer6_Neurons);
	hipFree(Vgg_Layer7_Neurons);
	hipFree(Vgg_Layer8_Neurons);
	hipFree(Vgg_Layer9_Neurons);
	hipFree(Vgg_Layer10_Neurons);
	hipFree(Vgg_Layer11_Neurons);
	hipFree(Vgg_Layer12_Neurons);
	hipFree(Vgg_Layer13_Neurons);
	hipFree(Vgg_Layer14_Neurons);
	hipFree(Vgg_Layer15_Neurons);
	hipFree(Vgg_Layer16_Neurons);

	hipFree(Vgg_Layer1_bias);
	hipFree(Vgg_Layer2_bias);
	hipFree(Vgg_Layer3_bias);
	hipFree(Vgg_Layer4_bias);
	hipFree(Vgg_Layer5_bias);
	hipFree(Vgg_Layer6_bias);
	hipFree(Vgg_Layer7_bias);
	hipFree(Vgg_Layer8_bias);
	hipFree(Vgg_Layer9_bias);
	hipFree(Vgg_Layer10_bias);
	hipFree(Vgg_Layer11_bias);
	hipFree(Vgg_Layer12_bias);
	hipFree(Vgg_Layer13_bias);
	hipFree(Vgg_Layer14_bias);
	hipFree(Vgg_Layer15_bias);
	hipFree(Vgg_Layer16_bias);

	hipFree(Vgg_Layer1_Weights);
	hipFree(Vgg_Layer2_Weights);
	hipFree(Vgg_Layer3_Weights);
	hipFree(Vgg_Layer4_Weights);
	hipFree(Vgg_Layer5_Weights);
	hipFree(Vgg_Layer6_Weights);
	hipFree(Vgg_Layer7_Weights);
	hipFree(Vgg_Layer8_Weights);
	hipFree(Vgg_Layer9_Weights);
	hipFree(Vgg_Layer10_Weights);
	hipFree(Vgg_Layer11_Weights);
	hipFree(Vgg_Layer12_Weights);
	hipFree(Vgg_Layer13_Weights);
	hipFree(Vgg_Layer14_Weights);
	hipFree(Vgg_Layer15_Weights);
	hipFree(Vgg_Layer16_Weights);

	hipFree(Vgg_Layer2_pool);
	hipFree(Vgg_Layer4_pool);
	hipFree(Vgg_Layer7_pool);
	hipFree(Vgg_Layer10_pool);
	hipFree(Vgg_Layer13_pool);
	hipFree(Vgg_Result_Neurons);
}
}




